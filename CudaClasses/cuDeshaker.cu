#include "hip/hip_runtime.h"
/*
 * This file is part of CUVISTA - Cuda Video Stabilizer
 * Copyright (c) 2023 Rainer Bitschi cuvista@a1.net
 *
 * This program is free software : you can redistribute it and /or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < http://www.gnu.org/licenses/>.
 */

#include "cuDeshaker.cuh"
#include "cuNPP.cuh"
#include "Image.hpp"

unsigned char* d_yuvData;			     //continuous array of all pixel values in yuv format, allocated on device
unsigned char** d_yuvRows;			     //index into rows of pixels, allocated on device
unsigned char*** d_yuvPlanes;		     //index into Y-U-V planes of frames, allocated on device 

unsigned char* d_yuvOut;   //image data for encoding on host
unsigned char* d_rgb;      //image data for progress update

float* d_bufferData;
std::vector<float*> bufferFrames; //index to one buffer frame, allocated on host

float* d_pyrData;
float** d_pyrRows;

//declare memory for index lookup during filter operations
__constant__ float constFilterKernels[] = { 0.0625f, 0.25f, 0.375f, 0.25f, 0.0625f, 0.25f, 0.5f, 0.25f, -0.5f, 0.0f, 0.5f };
float* filterKernelGauss[3];
int filterKernelGaussSizes[] = { 5, 3, 3 };
float* filterKernelDifference;

//results from compute kernel
PointResult* d_results;

//init cuda streams
std::vector<hipStream_t> cs(2);

//data output from kernels for later analysis
cu::DebugData debugData = {};

//registered memory
void* registeredMemPtr = nullptr;

//textures used in compute kernel
ComputeTextures compTex;

//array of time captures for compute kernel
KernelTimer* d_kernelTimer = nullptr;

//signal to interrupt compute kernel
char* d_interrupt;
//array to keep track of already computed blocks
char* d_computed;

__device__ void KernelTimer::start() {
	block = blockIdx;
	thread = threadIdx;
	cu::globaltimer(&timeStart);
}

__device__ void KernelTimer::stop() {
	cu::globaltimer(&timeStop);
}


//-------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------- HOST CODE ------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------

void handleStatus(hipError_t status, std::string&& title) {
	if (status != hipSuccess) {
		errorLogger.logError(title + ": " + hipGetErrorString(status));
	}
}

hipTextureObject_t prepareComputeTexture(float* src, int w, int h, int pitch) {
	hipResourceDesc resDesc {};
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = src;
	resDesc.res.pitch2D.width = w;
	resDesc.res.pitch2D.height = h;
	resDesc.res.pitch2D.pitchInBytes = pitch;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();

	// Specify texture object parameters
	hipTextureDesc texDesc {};
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;

	hipTextureObject_t texObj;
	handleStatus(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "error @compute 10");
	return texObj;
}

void ComputeTextures::create(int64_t idx, int64_t idxPrev, const CoreData& core) {
	size_t pyramidSize = 1ull * core.pyramidRows * core.strideCount; //size of one full pyramid in elements
	float* ptr1 = d_pyrData + 3 * pyramidSize * idx;
	Ycur = prepareComputeTexture(ptr1, core.w, core.pyramidRows, core.strideFloatBytes);

	float* ptr2 = d_pyrData + 3 * pyramidSize * idxPrev;
	Yprev = prepareComputeTexture(ptr2, core.w, core.pyramidRows, core.strideFloatBytes);
	DXprev = prepareComputeTexture(ptr2 + pyramidSize, core.w, core.pyramidRows, core.strideFloatBytes);
	DYprev = prepareComputeTexture(ptr2 + 2 * pyramidSize, core.w, core.pyramidRows, core.strideFloatBytes);
}

void ComputeTextures::destroy() {
	hipDestroyTextureObject(Ycur);
	hipDestroyTextureObject(Yprev);
	hipDestroyTextureObject(DXprev);
	hipDestroyTextureObject(DYprev);
}

//allocate cuda memory and store pointers
template <class T> void allocSafe(T* ptr, size_t size) {
	handleStatus(hipMalloc(ptr, size), "error @init allocating memory");
}

template <class T> void allocDeviceIndices(T*** indexArray, T* srcptr, size_t offset, size_t count) {
	std::vector<T*> idxarr(count);
	size_t siz = sizeof(T*) * count;
	for (size_t i = 0; i < count; i++) idxarr[i] = srcptr + i * offset;
	allocSafe(indexArray, siz);
	handleStatus(hipMemcpy(*indexArray, idxarr.data(), siz, hipMemcpyDefault), "error @init copy");
}

bool checkKernelParameters(int3 threads, int3 blocks, size_t shdsize, const hipDeviceProp_t& cudaProps) {
	bool out = true;
	out &= threads.x <= cudaProps.maxThreadsDim[0];
	out &= threads.y <= cudaProps.maxThreadsDim[1];
	out &= threads.z <= cudaProps.maxThreadsDim[2];
	out &= blocks.x <= cudaProps.maxGridSize[0];
	out &= blocks.y <= cudaProps.maxGridSize[1];
	out &= blocks.z <= cudaProps.maxGridSize[2];
	out &= shdsize <= cudaProps.sharedMemPerBlock;
	out &= threads.x * threads.y * threads.z <= cudaProps.maxThreadsPerBlock;
	return out;
}

bool checkKernelParameters(int3 threads, int3 blocks, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(threads, blocks, 0, cudaProps);
}

bool checkKernelParameters(const CoreData& core, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(core.computeThreads, core.computeBlocks, core.computeSharedMem, cudaProps);
}

//write data from device pointer to file for debugging
template <class T> void writeDeviceDataToFile(const T* devData, size_t h, size_t wCount, size_t strideCount, const std::string& path) {
	std::vector<T> hostData(h * wCount);
	hipMemcpy2D(hostData.data(), sizeof(T) * wCount, devData, sizeof(T) * strideCount, sizeof(T) * wCount, h, hipMemcpyDeviceToHost);
	std::ofstream file(path, std::ios::binary);
	file.write(reinterpret_cast<char*>(&h), sizeof(size_t));
	file.write(reinterpret_cast<char*>(&wCount), sizeof(size_t));
	size_t sizT = sizeof(T);
	file.write(reinterpret_cast<char*>(&sizT), sizeof(size_t));
	file.write(reinterpret_cast<char*>(hostData.data()), hostData.size() * sizeof(T));
}

//write string into image given by device pointer
void writeText(const std::string& text, int x0, int y0, int scaleX, int scaleY, float* deviceData, const CoreData& core) {
	//create Image<float>
	int imh = 10 * scaleY;
	int siz = imh * core.strideFloatBytes;
	ImagePlanar<float> im(imh, core.w, core.strideCount, 3);

	//copy three horizontal stripes into host memory
	for (size_t z = 0; z < 3; z++) {
		float* src = deviceData + (y0 + z * core.h) * core.strideCount;
		float* dst = im.plane(z);
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}

	//write text
	im.writeText(text, x0, 0, scaleX, scaleY, ColorNorm::WHITE, ColorNorm::BLACK); //write into host memory

	//copy YUV planes back into device memory
	for (size_t z = 0; z < 3; z++) {
		float* src = im.plane(z);
		float* dst = deviceData + (y0 + z * core.h) * core.strideCount;
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}
}


//----------------------------------
//-------- INIT
//----------------------------------

//check for cuda runtime installation, this only needs link to cudart_static.lib
std::vector<hipDeviceProp_t> cudaProbeRuntime(CudaInfo& cudaInfo) {
	//do not check hipError_t here, absence of cuda will report error "CUDA driver is insufficient for CUDA runtime version"
	hipRuntimeGetVersion(&cudaInfo.cudaRuntimeVersion);
	hipDriverGetVersion(&cudaInfo.cudaDriverVersion);

	//if we found a proper cuda installation, ask for list of devices
	int deviceCount = 0;
	std::vector<hipDeviceProp_t> props;
	if (cudaInfo.cudaRuntimeVersion > 0) {
		handleStatus(hipGetDeviceCount(&deviceCount), "error probing cuda devices");
		for (int i = 0; i < deviceCount; i++) {
			hipDeviceProp_t devProp;
			handleStatus(hipGetDeviceProperties(&devProp, i), "error getting device properties");
			props.push_back(devProp);
		}

		//query npp version numbers, this loads nvcuda.dll
		//const NppLibraryVersion* libVer = nppGetLibVersion(); //nppc.lib
		//cudaInfo.nppMajor = libVer->major;
		//cudaInfo.nppMinor = libVer->minor;
		//cudaInfo.nppBuild = libVer->build;
	}
	return props;
}

void cudaInit(CoreData& core, int devIdx, const hipDeviceProp_t& prop, ImageYuv& yuvFrame) {
	//pin memory of transfer object
	registeredMemPtr = yuvFrame.data();
	handleStatus(hipHostRegister(registeredMemPtr, yuvFrame.dataSizeInBytes(), hipHostRegisterDefault), "error @init #2");

	const size_t h = core.h;
	const size_t w = core.w;

	handleStatus(hipSetDevice(devIdx), "cannot set device");
	//hipFree(0); //seems necessary in order to get a context later;

	//sum up required shared memory for compute kernel
	int doublesCount = 0
		+ core.iw * core.iw * 6   //sd
		+ core.iw * core.iw * 1	  //delta
		+ 6 * 6		//S
		+ 6 * 6     //g
		+ 6 * 1		//Apiv
		+ 6 * 1		//b
		+ 6 * 1     //eta
		+ 3 * 3  	//wp
		+ 3 * 3     //dwp;
		;
	core.computeSharedMem = doublesCount * sizeof(double);

	//compute kernel configuration
	core.computeBlocks = { core.ixCount, core.iyCount };
	int rows = std::max(core.iw, 6);
	int ws = prop.warpSize;
	core.computeThreads = { ws / rows, rows };

	//allocate storage for compute timings
	handleStatus(hipMalloc(&d_kernelTimer, sizeof(KernelTimer) * core.computeBlocks.x * core.computeBlocks.y), "error @int #5");

	//get stride values for byte and float data
	void* d_ptr = nullptr;
	size_t pitchval;
	handleStatus(hipMallocPitch(&d_ptr, &pitchval, core.w, 1), "error @init 6 probing pitch value");
	handleStatus(hipFree(d_ptr), "error @init 7 freeing memory");

	handleStatus(hipMallocPitch(&d_ptr, &pitchval, core.w * sizeof(float), 1), "error @init 8 probing float stride value");
	core.strideFloatBytes = (int) pitchval;
	handleStatus(hipFree(d_ptr), "error @init 9 freeing memory");
	core.strideCount = core.strideFloatBytes / sizeof(float);
	core.cudapitch = (int) pitchval;

	//set memory limit
	size_t heap = 0;
	size_t heapRequired = 0;
	handleStatus(hipDeviceGetLimit(&heap, hipLimitMallocHeapSize), "error @init #10");

	size_t frameSize8 = 3ull * core.cudapitch * h;			//bytes for yuv444 images
	heapRequired += frameSize8 * core.bufferCount;		//yuv input storage
	heapRequired += frameSize8 * 2;						//yuv out
	heapRequired += 3ull * core.strideFloatBytes * h * (core.zMax + 1ull) * core.pyramidCount;		//pyramid mit Y, DX, DY
	heapRequired += 1ull * core.strideFloatBytes * h * core.BUFFER_COUNT;						    //output buffer in floats
	heapRequired += sizeof(PointResult) * core.resultCount;										    //array of results structure
	heapRequired += 10 * 1024 * 1024;

	if (heapRequired < heap)
		handleStatus(hipDeviceSetLimit(hipLimitMallocHeapSize, heapRequired), "error @init #20");

	size_t yuvRowCount = 3ull * h * core.bufferCount;
	size_t memtotal, memfree1, memfree2;
	handleStatus(hipMemGetInfo(&memfree1, &memtotal), "error @init #30");

	//allocate debug storage
	allocSafe(&debugData.d_data, debugData.maxSize);
	handleStatus(hipMemset(debugData.d_data, 0, debugData.maxSize), "error @init #32");

	//setup filter kernel pointers
	float* constKernels;
	float** symbolAddress = &constKernels;
	handleStatus(hipGetSymbolAddress((void**) (symbolAddress), constFilterKernels), "error @init #40");
	filterKernelGauss[0] = constKernels;
	filterKernelGauss[1] = constKernels + 5;
	filterKernelGauss[2] = constKernels + 5;
	filterKernelDifference = constKernels + 8;

	//allocate frameResult array on device
	allocSafe(&d_results, sizeof(PointResult) * core.resultCount);
	//allocate output yuv array
	allocSafe(&d_yuvOut, frameSize8);
	allocSafe(&d_rgb, 3ull * w * h);

	//allocate memory for yuv input data in char format [0..255]
	allocSafe(&d_yuvData, frameSize8 * core.bufferCount);
	allocDeviceIndices(&d_yuvRows, d_yuvData, core.cudapitch, yuvRowCount);
	allocDeviceIndices(&d_yuvPlanes, d_yuvRows, h, core.bufferCount * 3ull);

	//allocate several buffers in float [0..1]
	allocSafe(&d_bufferData, core.strideFloatBytes * core.BUFFER_COUNT * h);

	bufferFrames.resize(core.BUFFER_COUNT);
	for (size_t i = 0; i < bufferFrames.size(); i++) bufferFrames[i] = d_bufferData + core.strideCount * h * i;

	//initialize background color in output buffer
	size_t siz = w * sizeof(float);
	for (size_t i = 0; i < 3; i++) {
		std::vector<float> bg(w * h, core.bgcol_yuv.colors[i]);
		//buffer 6-7-8 static blank background
		handleStatus(hipMemcpy2D(bufferFrames[6 + i], core.strideFloatBytes, bg.data(), siz, siz, h, hipMemcpyHostToDevice), "error @init #60");
		//buffer 9-10-11 first initialize to background
		handleStatus(hipMemcpy2D(bufferFrames[9 + i], core.strideFloatBytes, bg.data(), siz, siz, h, hipMemcpyHostToDevice), "error @init #61");
	}

	//allocate image pyramids, all the same strided width but increasingly shorter
	//number of rows through all three pyramids, Y, DX, DY
	size_t pyrTotalRows = 3ull * core.pyramidRows * core.pyramidCount;
	allocSafe(&d_pyrData, core.strideFloatBytes * pyrTotalRows);
	allocDeviceIndices(&d_pyrRows, d_pyrData, core.strideCount, pyrTotalRows);

	//set up cuda streams
	for (size_t i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamCreate(&cs[i]), "error @init #70");
	}

	//set up compute kernel
	computeInit(core);
	allocSafe(&d_interrupt, 1);
	allocSafe(&d_computed, 1ll * core.ixCount * core.iyCount);

	//memory statistics
	handleStatus(hipMemGetInfo(&memfree2, &memtotal), "error @init #80");
	core.cudaMemTotal = memtotal;
	core.cudaUsedMem = memfree1 - memfree2;

	//final error checks
	handleStatus(hipDeviceSynchronize(), "error @init #90");
	handleStatus(hipGetLastError(), "error @init #92");
}


//----------------------------------
//-------- READ
//----------------------------------

//copy yuv input to device
void cudaReadFrame(int64_t frameIdx, const CoreData& core, const ImageYuv& inputFrame) {
	int64_t fr = frameIdx % core.bufferCount;
	size_t frameSizeBytes = 3ull * core.cudapitch * core.h;
	unsigned char* d_frame = d_yuvData + fr * frameSizeBytes;
	handleStatus(hipMemcpy2D(d_frame, core.cudapitch, inputFrame.data(), inputFrame.stride, core.w, 3ull * core.h, hipMemcpyDefault), "error @read copy");
	handleStatus(hipGetLastError(), "error @read unspecified");
}


//----------------------------------
//-------- PYRAMID
//----------------------------------

//create image pyramid for Y, DX, DY
void cudaCreatePyramid(int64_t frameIdx, const CoreData& core) {
	int w = core.w;
	int h = core.h;
	int64_t frIdx = frameIdx % core.bufferCount;
	unsigned char* yuvStart = d_yuvData + frIdx * core.cudapitch * h * 3; //get to the start of this yuv image

	int64_t pyrIdx = frameIdx % core.pyramidCount;
	float* pyrStart = d_pyrData + pyrIdx * core.pyramidRows * 3 * core.strideCount; //get to the start of this pyramid
	size_t planeOffset = 1ull * core.strideCount * core.pyramidRows;

	//first level of pyramid
	//Y data
	cu::scale_8u32f(yuvStart, core.cudapitch, pyrStart, core.strideCount, w, h);
	//DX data
	cu::filter_32f(pyrStart, pyrStart + planeOffset, core.strideFloatBytes, w, h, filterKernelDifference, 3, cu::FilterDim::FILTER_HORIZONZAL);
	//DY data
	cu::filter_32f(pyrStart, pyrStart + planeOffset * 2, core.strideFloatBytes, w, h, filterKernelDifference, 3, cu::FilterDim::FILTER_VERTICAL);

	//lower levels
	float* pyrNext = pyrStart + 1ull * core.strideCount * h;
	for (int z = 1; z <= core.zMax; z++) {
		cu::filter_32f(pyrStart, bufferFrames[13], core.strideFloatBytes, w, h, filterKernelGauss[0], filterKernelGaussSizes[0], cu::FilterDim::FILTER_HORIZONZAL);
		cu::filter_32f(bufferFrames[13], bufferFrames[12], core.strideFloatBytes, w, h, filterKernelGauss[0], filterKernelGaussSizes[0], cu::FilterDim::FILTER_VERTICAL);
		cu::remap_downsize_32f(bufferFrames[12], core.strideFloatBytes, pyrNext, core.strideCount, w, h);
		w /= 2;
		h /= 2;
		pyrStart = pyrNext;
		pyrNext += 1ull * core.strideCount * h;
		cu::filter_32f(pyrStart, pyrStart + planeOffset, core.strideFloatBytes, w, h, filterKernelDifference, 3, cu::FilterDim::FILTER_HORIZONZAL);
		cu::filter_32f(pyrStart, pyrStart + planeOffset * 2, core.strideFloatBytes, w, h, filterKernelDifference, 3, cu::FilterDim::FILTER_VERTICAL);
	}

	handleStatus(hipGetLastError(), "error @pyramid");
}


//----------------------------------
//-------- COMPUTE
//----------------------------------

void cudaCompute1(int64_t frameIdx, const CoreData& core, const hipDeviceProp_t& props) {
	int64_t pyrIdx = frameIdx % core.pyramidCount;
	size_t pyrIdxPrev = (pyrIdx == 0 ? core.pyramidCount : pyrIdx) - 1;

	assert(checkKernelParameters(core, props) && "invalid kernel parameters");
	compTex.create(pyrIdx, pyrIdxPrev, core);
	handleStatus(hipMemsetAsync(d_computed, 0, 1ll * core.ixCount * core.iyCount, cs[0]), "error @compute #20");
	ComputeKernelParam param = { 
		core.computeBlocks, 
		core.computeThreads, 
		core.computeSharedMem, 
		cs[0], 
		&debugData, 
		d_kernelTimer, 
		frameIdx, 
		d_interrupt, 
		d_computed 
	};
	kernelComputeCall(param, compTex, d_results);

	//hipStreamQuery(cs[0]);
	handleStatus(hipGetLastError(), "error @compute #20");
}

void cudaCompute2(int64_t frameIdx, const CoreData& core) {
	//reset interrupt signal
	handleStatus(hipMemsetAsync(d_interrupt, 0, sizeof(char), cs[1]), "error @compute #50");
	//restart kernel
	ComputeKernelParam param = {
		core.computeBlocks,
		core.computeThreads,
		core.computeSharedMem,
		cs[0],
		&debugData,
		d_kernelTimer,
		frameIdx,
		d_interrupt,
		d_computed
	};
	kernelComputeCall(param, compTex, d_results);
	handleStatus(hipGetLastError(), "error @compute #30");
}

void cudaComputeTerminate(const CoreData& core, std::vector<PointResult>& results) {
	//handleStatus(hipMemcpyAsync(results.data(), d_results, sizeof(PointResult) * results.size(), hipMemcpyDefault, cs1), "error @compute #40", err);
	handleStatus(hipMemcpy(results.data(), d_results, sizeof(PointResult) * results.size(), hipMemcpyDefault), "error @compute #100");
	compTex.destroy();
	handleStatus(hipGetLastError(), "error @compute #100");
}


//----------------------------------
//-------- OUTPUT
//----------------------------------

/*
buffer frames usage, each frame holds one plane Y, U, V in float format
	0,  1,  2: input converted from yuv
	3,  4,  5: start with background, then put warped data from input there
	6,  7,  8: background color
	9, 10, 11: background to use for blending next frame
	12, 13, 14: temporary buffer for gauss filtering
	15, 16, 17: gauss filter result
	18, 19, 20: output to encoder
*/
void cudaOutput(int64_t frameIdx, const CoreData& core, OutputContext outCtx, cu::Affine trf) {
	//interrupt compute kernel
	handleStatus(hipMemsetAsync(d_interrupt, 1, sizeof(char), cs[1]), "error @output #10 memset");

	int h = core.h;
	int w = core.w;
	int64_t fr = frameIdx % core.bufferCount;

	//size of all pixel data in bytes in yuv including padding
	size_t frameSize8 = 3ull * core.cudapitch * h;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * frameSize8;

	//handle three frames at once, convert input image from 8bit to float
	cu::scale_8u32f(yuvSrc, core.cudapitch, bufferFrames[0], core.strideCount, w, h * 3, cs[1]);

	//handle individual frames
	const BlendInput& bi = core.blendInput;
	for (size_t i = 0; i < 3; i++) {
		float* in = bufferFrames[i];
		float* bg = bufferFrames[6 + i];
		float* warped = bufferFrames[9 + i];
		float* temp = bufferFrames[12 + i];
		float* buffer = bufferFrames[15 + i];
		float* out = bufferFrames[18 + i];

		//reset background with static color if requested
		if (core.bgmode == BackgroundMode::COLOR) {
			cu::copy_32f(bg, core.strideFloatBytes, warped, core.strideCount, w, h);
		}
		//transform input on top of background
		cu::warp_back_32f(in, core.strideFloatBytes, warped, core.strideCount, w, h, trf, cs[1]);
		//first filter pass
		cu::filter_32f(warped, temp, core.strideFloatBytes, w, h, filterKernelGauss[i], filterKernelGaussSizes[i], cu::FilterDim::FILTER_HORIZONZAL, cs[1]);
		//second filter pass
		cu::filter_32f(temp, buffer, core.strideFloatBytes, w, h, filterKernelGauss[i], filterKernelGaussSizes[i], cu::FilterDim::FILTER_VERTICAL, cs[1]);
		//combine unsharp mask
		cu::unsharp_32f(warped, buffer, core.strideFloatBytes, out, core.strideCount, w, h, core.unsharp[i], cs[1]);

		//blend input frame on top of output when requested
		if (bi.blendWidth > 0) {
			cu::copy_32f(in + bi.blendStart, core.strideFloatBytes, out + bi.blendStart, core.strideCount, bi.blendWidth, h);
			cu::copy_32f(bg + bi.separatorStart, core.strideFloatBytes, out + bi.separatorStart, core.strideCount, bi.separatorWidth, h);
		}
	}

	//if (frameIdx < 10) {
	//	writeDeviceDataToFile(bufferFrames[0], h, w, core.strideCount, "d:/inY-" + std::to_string(frameIdx) + ".dat");
	//	writeDeviceDataToFile(bufferFrames[9], h, w, core.strideCount, "d:/outY-" + std::to_string(frameIdx) + ".dat");
	//}

	//writeText(std::to_string(frameIdx), 10, 10, 2, 3, bufferFrames[18], core);

	//output to nvenc buffer
	if (outCtx.encodeCuda) {
		//convert Y plane
		cu::scale_32f8u(bufferFrames[18], core.strideFloatBytes, outCtx.cudaNv12ptr, outCtx.cudaPitch, w, h, cs[1]);
		//convert and interleave U and V plane
		cu::uv_to_nv12(bufferFrames[19], core.strideFloatBytes, outCtx.cudaNv12ptr + 1ull * outCtx.cudaPitch * h, outCtx.cudaPitch, w, h, cs[1]);
	}

	//output to host
	if (outCtx.encodeCpu) {
		cu::scale_32f8u(bufferFrames[18], core.strideFloatBytes, d_yuvOut, core.cudapitch, w, h * 3, cs[1]);
		ImageYuv* im = outCtx.outputFrame;
		handleStatus(hipMemcpy2D(im->data(), im->stride, d_yuvOut, core.cudapitch, w, 3ull * h, hipMemcpyDeviceToHost), "error @output #50 memcopy");
		outCtx.outputFrame->frameIdx = frameIdx;
	}

	handleStatus(hipStreamSynchronize(cs[1]), "error @output #99");
	handleStatus(hipGetLastError(), "error @output #100");
}

void encodeNvData(const std::vector<unsigned char>& nv12, unsigned char* nvencPtr) {
	handleStatus(hipMemcpy(nvencPtr, nv12.data(), nv12.size(), hipMemcpyHostToDevice), "error @simple encode #1 cannot copy to device");
}

void getNvData(std::vector<unsigned char>& nv12, OutputContext outCtx) {
	handleStatus(hipMemcpy(nv12.data(), outCtx.cudaNv12ptr, nv12.size(), hipMemcpyDeviceToHost), "error getting nv12 data");
}


void cudaGetTransformedOutput(float* warpedData, const CoreData& core) {
	size_t width = core.w * sizeof(float);
	hipMemcpy2D(warpedData, width, bufferFrames[9], core.strideFloatBytes, width, core.h * 3ll, hipMemcpyDefault);
}

void cudaGetPyramid(float* pyramid, size_t idx, const CoreData& core) {
	size_t pyrIdx = idx % core.pyramidCount;
	float* devptr = d_pyrData + pyrIdx * core.pyramidRows * 3 * core.strideCount;
	size_t wbytes = core.w * sizeof(float);
	hipMemcpy2D(pyramid, wbytes, devptr, core.strideFloatBytes, wbytes, core.pyramidRows * 3ull, hipMemcpyDefault);
}

ImageYuv cudaGetInput(int64_t index, const CoreData& core) {
	ImageYuv out(core.h, core.w, core.w);
	int64_t fr = index % core.bufferCount;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * 3 * core.h * core.cudapitch;
	//copy 2D data without stride
	hipMemcpy2D(out.data(), out.w, yuvSrc, core.cudapitch, out.w, 3ll * out.h, hipMemcpyDefault);
	return out;
}

void cudaGetCurrentInputFrame(ImagePPM& image, const CoreData& core, int idx) {
	unsigned char* yuvSrc = d_yuvData + idx * 3ll * core.h * core.cudapitch;
	cu::yuv_to_rgb(yuvSrc, core.cudapitch, d_rgb, core.strideCount, core.w, core.h);
	hipMemcpy2D(image.data(), image.w, d_rgb, core.cudapitch, core.w, 3ull * core.h, hipMemcpyDefault);
}

void cudaGetCurrentOutputFrame(ImagePPM& image, const CoreData& core) {
	cu::yuv_to_rgb(bufferFrames[9], core.strideFloatBytes, d_rgb, core.strideCount, core.w, core.h);
	hipMemcpy2D(image.data(), image.w, d_rgb, core.cudapitch, core.w, 3ull * core.h, hipMemcpyDefault);
}


//----------------------------------
//-------- SYNCHRONIZE
//----------------------------------

void cudaSynchronize() {
	handleStatus(hipDeviceSynchronize(), "error @synchronize #10");
	handleStatus(hipGetLastError(), "error @synchronize #20");
}


//----------------------------------
//-------- SHUTDOWN
//----------------------------------

DebugData cudaShutdown(const CoreData& core) {
	//get debug data
	std::vector<double> outDebug(debugData.maxSize / sizeof(double));
	handleStatus(hipMemcpy(outDebug.data(), debugData.d_data, debugData.maxSize, hipMemcpyDeviceToHost), "error @shutdown #5 copy debug data");

	//get image of kernel timing values
	int siz = core.computeBlocks.x * core.computeBlocks.y;
	std::vector<KernelTimer> kernelTimer(siz);
	int h = (int) kernelTimer.size();
	int w = 8'000;
	ImageBGR kernelTimerImage(h, w);
	
	handleStatus(hipMemcpy(kernelTimer.data(), d_kernelTimer, sizeof(KernelTimer) * kernelTimer.size(), hipMemcpyDefault), "error @shutdown #110");

	auto fcnMin = [] (KernelTimer& kt1, KernelTimer& kt2) { return kt1.timeStart < kt2.timeStart; };
	auto minTime = std::min_element(kernelTimer.begin(), kernelTimer.end(), fcnMin);
	auto fcnMax = [] (KernelTimer& kt1, KernelTimer& kt2) { return kt1.timeStop < kt2.timeStop; };
	auto maxTime = std::max_element(kernelTimer.begin(), kernelTimer.end(), fcnMax);
	double f = (maxTime->timeStop - minTime->timeStart) / (w - 1.0);

	for (int i = 0; i < core.computeBlocks.x * core.computeBlocks.y; i++) {
		KernelTimer& kt = kernelTimer[i];
		int t1 = int((kt.timeStart - minTime->timeStart) / f);
		int t2 = int((kt.timeStop - minTime->timeStart) / f);
		for (int k = t1; k <= t2; k++) {
			kernelTimerImage.at(0, i, k) = 255;
		}
	}
	
	//delete device memory
	void* d_arr[] = { d_results, d_yuvOut, d_rgb, d_yuvData, d_yuvRows, d_yuvPlanes, 
		d_bufferData, d_pyrData, d_pyrRows, debugData.d_data, d_kernelTimer, d_interrupt, d_computed };

	for (void* ptr : d_arr) {
		handleStatus(hipFree(ptr), "error @shutdown #10 shutting down memory");
	}

	//delete streams
	for (int i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamDestroy(cs[i]), "error @shutdown #20 shutting down streams");
	}

	//unregister memory
	handleStatus(hipHostUnregister(registeredMemPtr), "error @shutdown #30 unregister");

	//do not reset device while nvenc is still active
	//handleStatus(hipDeviceReset(), "error @shutdown #90", errorList);
	handleStatus(hipGetLastError(), "error @shutdown #100");

	return { outDebug, kernelTimerImage };
}
