#include "hip/hip_runtime.h"
/*
 * This file is part of CUVISTA - Cuda Video Stabilizer
 * Copyright (c) 2023 Rainer Bitschi cuvista@a1.net
 *
 * This program is free software : you can redistribute it and /or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < http://www.gnu.org/licenses/>.
 */

#include "cuDeshaker.cuh"
#include "cuNPP.cuh"
#include "Image.hpp"

unsigned char* d_yuvData;			     //continuous array of all pixel values in yuv format, allocated on device
unsigned char** d_yuvRows;			     //index into rows of pixels, allocated on device
unsigned char*** d_yuvPlanes;		     //index into Y-U-V planes of frames, allocated on device 

unsigned char* d_yuvOut;   //image data for encoding on host
unsigned char* d_rgb;      //image data for progress update

struct {
	float4* data;

	float4* start;
	float4* warped;
	float4* filterH;
	float4* filterV;
	float4* final;
	float4* background;
} out;

float* d_bufferH;
float* d_bufferV;

float* d_pyrData;
float** d_pyrRows;

//results from compute kernel
PointResult* d_results;

//init cuda streams
std::vector<hipStream_t> cs(2);

//data output from kernels for later analysis
cu::DebugData debugData = {};

//registered memory
void* registeredMemPtr = nullptr;

//textures used in compute kernel
ComputeTextures compTex;

//array of time captures for compute kernel
KernelTimer* d_kernelTimer = nullptr;

//signal to interrupt compute kernel
char* d_interrupt;
//array to keep track of already computed blocks
char* d_computed;

//parameter structure
__constant__ CudaData d_core;


__device__ void KernelTimer::start() {
	block = blockIdx;
	thread = threadIdx;
	cu::globaltimer(&timeStart);
}

__device__ void KernelTimer::stop() {
	cu::globaltimer(&timeStop);
}

__device__ const CudaData& getCudaData() {
	return d_core;
}

//-------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------- HOST CODE ------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------

void handleStatus(hipError_t status, std::string&& title) {
	if (status != hipSuccess) {
		errorLogger.logError(title + ": " + hipGetErrorString(status));
	}
}

hipTextureObject_t prepareComputeTexture(float* src, int w, int h, int pitch) {
	hipResourceDesc resDesc {};
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = src;
	resDesc.res.pitch2D.width = w;
	resDesc.res.pitch2D.height = h;
	resDesc.res.pitch2D.pitchInBytes = pitch;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();

	// Specify texture object parameters
	hipTextureDesc texDesc {};
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;

	hipTextureObject_t texObj;
	handleStatus(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "error @compute 10");
	return texObj;
}

void ComputeTextures::create(int64_t idx, int64_t idxPrev, const CudaData& core) {
	size_t pyramidSize = 1ull * core.pyramidRowCount * core.strideFloatN; //size of one full pyramid in elements
	float* ptr1 = d_pyrData + 3 * pyramidSize * idx;
	Ycur = prepareComputeTexture(ptr1, core.w, core.pyramidRowCount, core.strideFloat);

	float* ptr2 = d_pyrData + 3 * pyramidSize * idxPrev;
	Yprev = prepareComputeTexture(ptr2, core.w, core.pyramidRowCount, core.strideFloat);
	DXprev = prepareComputeTexture(ptr2 + pyramidSize, core.w, core.pyramidRowCount, core.strideFloat);
	DYprev = prepareComputeTexture(ptr2 + 2 * pyramidSize, core.w, core.pyramidRowCount, core.strideFloat);
}

void ComputeTextures::destroy() {
	hipDestroyTextureObject(Ycur);
	hipDestroyTextureObject(Yprev);
	hipDestroyTextureObject(DXprev);
	hipDestroyTextureObject(DYprev);
}

//allocate cuda memory and store pointers
template <class T> void allocSafe(T* ptr, size_t size) {
	handleStatus(hipMalloc(ptr, size), "error @init allocating memory");
}

template <class T> void allocDeviceIndices(T*** indexArray, T* srcptr, size_t offset, size_t count) {
	std::vector<T*> idxarr(count);
	size_t siz = sizeof(T*) * count;
	for (size_t i = 0; i < count; i++) idxarr[i] = srcptr + i * offset;
	allocSafe(indexArray, siz);
	handleStatus(hipMemcpy(*indexArray, idxarr.data(), siz, hipMemcpyDefault), "error @init copy");
}

bool checkKernelParameters(int3 threads, int3 blocks, size_t shdsize, const hipDeviceProp_t& cudaProps) {
	bool out = true;
	out &= threads.x <= cudaProps.maxThreadsDim[0];
	out &= threads.y <= cudaProps.maxThreadsDim[1];
	out &= threads.z <= cudaProps.maxThreadsDim[2];
	out &= blocks.x <= cudaProps.maxGridSize[0];
	out &= blocks.y <= cudaProps.maxGridSize[1];
	out &= blocks.z <= cudaProps.maxGridSize[2];
	out &= shdsize <= cudaProps.sharedMemPerBlock;
	out &= threads.x * threads.y * threads.z <= cudaProps.maxThreadsPerBlock;
	return out;
}

bool checkKernelParameters(int3 threads, int3 blocks, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(threads, blocks, 0, cudaProps);
}

bool checkKernelParameters(const CudaData& core, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(core.computeThreads, core.computeBlocks, core.computeSharedMem, cudaProps);
}

//write data from device pointer to file for debugging
template <class T> void writeDeviceDataToFile(const T* devData, size_t h, size_t wCount, size_t strideFloatN, const std::string& path) {
	std::vector<T> hostData(h * wCount);
	hipMemcpy2D(hostData.data(), sizeof(T) * wCount, devData, sizeof(T) * strideFloatN, sizeof(T) * wCount, h, hipMemcpyDeviceToHost);
	std::ofstream file(path, std::ios::binary);
	file.write(reinterpret_cast<char*>(&h), sizeof(size_t));
	file.write(reinterpret_cast<char*>(&wCount), sizeof(size_t));
	size_t sizT = sizeof(T);
	file.write(reinterpret_cast<char*>(&sizT), sizeof(size_t));
	file.write(reinterpret_cast<char*>(hostData.data()), hostData.size() * sizeof(T));
}

//write string into image given by device pointer
void writeText(const std::string& text, int x0, int y0, int scaleX, int scaleY, float* deviceData, const CudaData& core) {
	//create Image<float>
	int imh = 10 * scaleY;
	int siz = imh * core.strideFloat;
	ImagePlanar<float> im(imh, core.w, core.strideFloatN, 3);

	//copy three horizontal stripes into host memory
	for (size_t z = 0; z < 3; z++) {
		float* src = deviceData + (y0 + z * core.h) * core.strideFloatN;
		float* dst = im.plane(z);
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}

	//write text
	im.writeText(text, x0, 0, scaleX, scaleY, ColorNorm::WHITE, ColorNorm::BLACK); //write into host memory

	//copy YUV planes back into device memory
	for (size_t z = 0; z < 3; z++) {
		float* src = im.plane(z);
		float* dst = deviceData + (y0 + z * core.h) * core.strideFloatN;
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}
}

int align(size_t base, size_t alignment) {
	return (int) ((base + alignment - 1) / alignment * alignment);
}


//----------------------------------
//-------- INIT
//----------------------------------

//check for cuda runtime installation, this only needs link to cudart_static.lib
std::vector<hipDeviceProp_t> cudaProbeRuntime(CudaInfo& cudaInfo) {
	//do not check hipError_t here, absence of cuda will report error "CUDA driver is insufficient for CUDA runtime version"
	hipRuntimeGetVersion(&cudaInfo.cudaRuntimeVersion);
	hipDriverGetVersion(&cudaInfo.cudaDriverVersion);

	//if we found a proper cuda installation, ask for list of devices
	int deviceCount = 0;
	std::vector<hipDeviceProp_t> props;
	if (cudaInfo.cudaRuntimeVersion > 0) {
		handleStatus(hipGetDeviceCount(&deviceCount), "error probing cuda devices");
		for (int i = 0; i < deviceCount; i++) {
			hipDeviceProp_t devProp;
			handleStatus(hipGetDeviceProperties(&devProp, i), "error getting device properties");
			props.push_back(devProp);
		}

		//query npp version numbers, this loads nvcuda.dll
		//const NppLibraryVersion* libVer = nppGetLibVersion(); //nppc.lib
		//cudaInfo.nppMajor = libVer->major;
		//cudaInfo.nppMinor = libVer->minor;
		//cudaInfo.nppBuild = libVer->build;
	}
	return props;
}

void cudaInit(CudaData& core, int devIdx, const hipDeviceProp_t& prop, ImageYuv& yuvFrame) {
	//pin memory of transfer object
	registeredMemPtr = yuvFrame.data();
	handleStatus(hipHostRegister(registeredMemPtr, yuvFrame.dataSizeInBytes(), hipHostRegisterDefault), "error @init #2");

	const size_t h = core.h;
	const size_t w = core.w;

	handleStatus(hipSetDevice(devIdx), "cannot set device");
	//hipFree(0); //seems necessary in order to get a context later;

	//sum up required shared memory for compute kernel
	int doublesCount = 0
		+ core.iw * core.iw * 6   //sd
		+ core.iw * core.iw * 1	  //delta
		+ 6 * 6		//S
		+ 6 * 6     //g
		+ 6 * 1		//Apiv
		+ 6 * 1		//b
		+ 6 * 1     //eta
		+ 3 * 3  	//wp
		+ 3 * 3     //dwp;
		;
	core.computeSharedMem = doublesCount * sizeof(double);

	//compute kernel configuration
	core.computeBlocks = { core.ixCount, core.iyCount };
	int rows = std::max(core.iw, 6);
	int ws = prop.warpSize;
	core.computeThreads = { ws / rows, rows };

	//allocate storage for compute timings
	handleStatus(hipMalloc(&d_kernelTimer, sizeof(KernelTimer) * core.computeBlocks.x * core.computeBlocks.y), "error @int #5");
	
	//determine memory requirements
	//size_t texAlign = prop.texturePitchAlignment;
	size_t pitch = 0;
	uchar* d_ptr8;
	hipMallocPitch(&d_ptr8, &pitch, w, 1);
	core.strideChar = (int) pitch;
	hipFree(d_ptr8);

	float* d_ptr32;
	hipMallocPitch(&d_ptr32, &pitch, w * sizeof(float), 1);
	core.strideFloat = (int) pitch;
	core.strideFloatN = core.strideFloat / sizeof(float);
	hipFree(d_ptr32);

	float4* d_ptr128;
	hipMallocPitch(&d_ptr128, &pitch, w * sizeof(float4), 1);
	core.strideFloat4 = (int) pitch;
	core.strideFloat4N = core.strideFloat4 / sizeof(float4);
	hipFree(d_ptr128);

	//compute required heap size
	size_t frameSize8 = 3ull * core.strideChar * h;		//bytes for yuv444 images
	size_t heapRequired = 0;
	heapRequired += frameSize8 * core.bufferCount;		//yuv input storage
	heapRequired += frameSize8;						    //yuv out
	heapRequired += frameSize8;						    //rgb out
	heapRequired += 2ull * core.strideFloat * h;        //filter buffers
	heapRequired += 3ull * core.strideFloat * h * core.pyramidLevels * core.pyramidCount;		//pyramid mit Y, DX, DY
	heapRequired += 1ull * core.strideFloat4 * h * core.outBufferCount;						    //output buffer in floats
	heapRequired += sizeof(PointResult) * core.resultCount;									    //array of results structure
	heapRequired += 10 * 1024 * 1024;

	//set memory limit
	size_t heap = 0;
	handleStatus(hipDeviceGetLimit(&heap, hipLimitMallocHeapSize), "error @init #10");
	if (heapRequired < heap) {
		handleStatus(hipDeviceSetLimit(hipLimitMallocHeapSize, heapRequired), "error @init #20");
	}

	size_t yuvRowCount = 3ull * h * core.bufferCount;
	size_t memtotal, memfree1, memfree2;
	handleStatus(hipMemGetInfo(&memfree1, &memtotal), "error @init #30");

	//allocate debug storage
	allocSafe(&debugData.d_data, debugData.maxSize);
	handleStatus(hipMemset(debugData.d_data, 0, debugData.maxSize), "error @init #32");

	//allocate frameResult array on device
	allocSafe(&d_results, sizeof(PointResult) * core.resultCount);
	//allocate output yuv array
	allocSafe(&d_yuvOut, frameSize8);
	allocSafe(&d_rgb, 3ull * w * h);

	//allocate memory for yuv input data in char format [0..255]
	allocSafe(&d_yuvData, frameSize8 * core.bufferCount);
	allocDeviceIndices(&d_yuvRows, d_yuvData, core.strideChar, yuvRowCount);
	allocDeviceIndices(&d_yuvPlanes, d_yuvRows, h, core.bufferCount * 3ull);

	//allocate float buffers
	allocSafe(&out.data, core.strideFloat4 * h * core.outBufferCount);
	//name individual parts for convenience
	size_t outSize = h * core.strideFloat4N;
	out.start = out.data;
	out.warped = out.start + outSize;
	out.filterH = out.warped + outSize;
	out.filterV = out.filterH + outSize;
	out.final = out.filterV + outSize;
	out.background = out.final + outSize;

	//float filter buffers
	allocSafe(&d_bufferH, core.strideFloat * h);
	allocSafe(&d_bufferV, core.strideFloat * h);

	//initialize background color in output buffer
	float4 bgval = { core.bgcol_yuv.colors[0], core.bgcol_yuv.colors[1], core.bgcol_yuv.colors[2] };
	std::vector<float4> bg(w * h, bgval);
	//write to static background
	size_t siz = w * sizeof(float4);
	handleStatus(hipMemcpy2D(out.background, core.strideFloat4, bg.data(), siz, siz, h, hipMemcpyDefault), "error @init 60");
	//write to first image
	handleStatus(hipMemcpy2D(out.warped, core.strideFloat4, bg.data(), siz, siz, h, hipMemcpyDefault), "error @init 61");

	//allocate image pyramids, all the same strided width but increasingly shorter
	//number of rows through all three pyramids, Y, DX, DY
	size_t pyrTotalRows = 3ull * core.pyramidRowCount * core.pyramidCount;
	allocSafe(&d_pyrData, core.strideFloat * pyrTotalRows);
	allocDeviceIndices(&d_pyrRows, d_pyrData, core.strideFloatN, pyrTotalRows);

	//set up cuda streams
	for (size_t i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamCreate(&cs[i]), "error @init #70");
	}

	//set up compute kernel
	allocSafe(&d_interrupt, 1);
	allocSafe(&d_computed, 1ll * core.ixCount * core.iyCount);

	//memory statistics
	handleStatus(hipMemGetInfo(&memfree2, &memtotal), "error @init #80");
	core.cudaMemTotal = memtotal;
	core.cudaUsedMem = memfree1 - memfree2;

	//copy core struct to device
	const void* coreptr = &d_core;
	hipMemcpyToSymbol(HIP_SYMBOL(coreptr), &core, sizeof(core));

	//final error checks
	handleStatus(hipDeviceSynchronize(), "error @init #90");
	handleStatus(hipGetLastError(), "error @init #92");
}


//----------------------------------
//-------- READ
//----------------------------------

//copy yuv input to device
void cudaReadFrame(int64_t frameIdx, const CudaData& core, const ImageYuv& inputFrame) {
	int64_t fr = frameIdx % core.bufferCount;
	size_t frameSizeBytes = 3ull * core.strideChar * core.h;
	unsigned char* d_frame = d_yuvData + fr * frameSizeBytes;
	handleStatus(hipMemcpy2D(d_frame, core.strideChar, inputFrame.data(), inputFrame.stride, core.w, 3ull * core.h, hipMemcpyDefault), "error @read #10");
	handleStatus(hipGetLastError(), "error @read #20");
}


//----------------------------------
//-------- PYRAMID
//----------------------------------

//create image pyramid for Y, DX, DY
void cudaCreatePyramid(int64_t frameIdx, const CudaData& core) {
	int w = core.w;
	int h = core.h;
	int64_t frIdx = frameIdx % core.bufferCount;
	unsigned char* yuvStart = d_yuvData + frIdx * core.strideChar * h * 3; //get to the start of this yuv image

	int64_t pyrIdx = frameIdx % core.pyramidCount;
	float* pyrStart = d_pyrData + pyrIdx * core.pyramidRowCount * 3 * core.strideFloatN; //get to the start of this pyramid
	size_t planeOffset = 1ull * core.strideFloatN * core.pyramidRowCount;

	//first level of pyramid
	//Y data
	cu::scale_8u32f(yuvStart, core.strideChar, pyrStart, core.strideFloatN, w, h);
	//DX data
	cu::filter_32f_h(pyrStart, pyrStart + planeOffset, core.strideFloatN, w, h, 3);
	//DY data
	cu::filter_32f_v(pyrStart, pyrStart + planeOffset * 2, core.strideFloatN, w, h, 3);

	//lower levels
	float* pyrNext = pyrStart + 1ull * core.strideFloatN * h;
	for (int z = 1; z <= core.zMax; z++) {
		cu::filter_32f_h(pyrStart, d_bufferH, core.strideFloatN, w, h, 0);
		cu::filter_32f_v(d_bufferH, d_bufferV, core.strideFloatN, w, h, 0);
		cu::remap_downsize_32f(d_bufferV, core.strideFloatN, pyrNext, core.strideFloatN, w, h);
		w /= 2;
		h /= 2;
		pyrStart = pyrNext;
		pyrNext += 1ull * core.strideFloatN * h;
		//DX
		cu::filter_32f_h(pyrStart, pyrStart + planeOffset, core.strideFloatN, w, h, 3);
		//DY
		cu::filter_32f_v(pyrStart, pyrStart + planeOffset * 2, core.strideFloatN, w, h, 3);
	}

	handleStatus(hipGetLastError(), "error @pyramid");
}


//----------------------------------
//-------- COMPUTE
//----------------------------------

void cudaCompute1(int64_t frameIdx, const CudaData& core, const hipDeviceProp_t& props) {
	assert(frameIdx > 0 && "invalid pyramid index");
	int64_t pyrIdx = frameIdx % core.pyramidCount;
	int64_t pyrIdxPrev = (frameIdx - 1) % core.pyramidCount;

	assert(checkKernelParameters(core, props) && "invalid kernel parameters");
	compTex.create(pyrIdx, pyrIdxPrev, core);
	//reset computed flags
	handleStatus(hipMemsetAsync(d_computed, 0, 1ll * core.ixCount * core.iyCount, cs[0]), "error @compute #20");
	ComputeKernelParam param = { 
		core.computeBlocks, 
		core.computeThreads, 
		core.computeSharedMem, 
		cs[0], 
		&debugData, 
		d_kernelTimer, 
		frameIdx, 
		d_interrupt, 
		d_computed 
	};
	kernelComputeCall(param, compTex, d_results);

	//hipStreamQuery(cs[0]);
	handleStatus(hipGetLastError(), "error @compute #20");
}

void cudaCompute2(int64_t frameIdx, const CudaData& core) {
	//reset interrupt signal
	handleStatus(hipMemsetAsync(d_interrupt, 0, sizeof(char), cs[1]), "error @compute #50");
	//restart kernel
	ComputeKernelParam param = {
		core.computeBlocks,
		core.computeThreads,
		core.computeSharedMem,
		cs[0],
		&debugData,
		d_kernelTimer,
		frameIdx,
		d_interrupt,
		d_computed
	};
	kernelComputeCall(param, compTex, d_results);
	handleStatus(hipGetLastError(), "error @compute #30");
}

void cudaComputeTerminate(const CudaData& core, std::vector<PointResult>& results) {
	//handleStatus(hipMemcpyAsync(results.data(), d_results, sizeof(PointResult) * results.size(), hipMemcpyDefault, cs1), "error @compute #40", err);
	handleStatus(hipMemcpy(results.data(), d_results, sizeof(PointResult) * results.size(), hipMemcpyDefault), "error @compute #100");
	compTex.destroy();
	handleStatus(hipGetLastError(), "error @compute #100");
}


//----------------------------------
//-------- OUTPUT
//----------------------------------

void cudaOutput(int64_t frameIdx, const CudaData& core, OutputContext outCtx, std::array<double, 6> trf) {
	//ConsoleTimer timer;
	//interrupt compute kernel
	handleStatus(hipMemsetAsync(d_interrupt, 1, sizeof(char), cs[1]), "error @output #10");

	int h = core.h;
	int w = core.w;
	int64_t fr = frameIdx % core.bufferCount;

	//size of all pixel data in bytes in yuv including padding
	size_t frameSize8 = 3ull * core.strideChar * h;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * frameSize8;

	cu::scale_8u32f_3(yuvSrc, core.strideChar, out.start, core.strideFloat4N, w, h, cs[1]);
	//fill static background when requested
	if (core.bgmode == BackgroundMode::COLOR) {
		cu::copy_32f_3(out.background, core.strideFloat4N, out.warped, core.strideFloat4N, w, h, cs[1]);
	}
	//warp input
	cu::Affine cutrf = { trf[0], trf[1], trf[2], trf[3], trf[4], trf[5] };
	cu::warp_back_32f_3(out.start, core.strideFloat4N, out.warped, core.strideFloat4N, w, h, cutrf, cs[1]);
	//first filter pass
	cu::filter_32f_h_3(out.warped, out.filterH, core.strideFloat4N, w, h, cs[1]);
	//second filter pass
	cu::filter_32f_v_3(out.filterH, out.filterV, core.strideFloat4N, w, h, cs[1]);
	//combine unsharp mask
	cu::unsharp_32f_3(out.warped, out.filterV, out.final, core.strideFloat4N, w, h, cs[1]);
	//blend input frame on top of output when requested
	const BlendInput& bi = core.blendInput;
	if (bi.blendWidth > 0) {
		cu::copy_32f_3(out.start + bi.blendStart, core.strideFloat4N, out.final + bi.blendStart, core.strideFloat4N, bi.blendWidth, h, cs[1]);
		cu::copy_32f_3(out.background + bi.separatorStart, core.strideFloat4N, out.final + bi.separatorStart, core.strideFloat4N, bi.separatorWidth, h, cs[1]);
	}
	//output to host
	if (outCtx.encodeCpu) {
		cu::outputHost(out.final, core.strideFloat4N, d_yuvOut, core.strideChar, w, h, cs[1]);
		ImageYuv* im = outCtx.outputFrame;
		cu::copy_32f_3(d_yuvOut, core.strideChar, im->data(), im->stride, w, h * 3, cs[1]);
		outCtx.outputFrame->frameIdx = frameIdx;
	}
	//output to nvenc
	if (outCtx.encodeCuda) {
		cu::outputNvenc(out.final, core.strideFloat4N, outCtx.cudaNv12ptr, outCtx.cudaPitch, w, h, cs[1]);
	}

	//writeText(std::to_string(frameIdx), 10, 10, 2, 3, bufferFrames[18], core);

	handleStatus(hipStreamSynchronize(cs[1]), "error @output #99");
	handleStatus(hipGetLastError(), "error @output #100");
}

void encodeNvData(const std::vector<unsigned char>& nv12, unsigned char* nvencPtr) {
	handleStatus(hipMemcpy(nvencPtr, nv12.data(), nv12.size(), hipMemcpyHostToDevice), "error @simple encode #1 cannot copy to device");
}

void getNvData(std::vector<unsigned char>& nv12, OutputContext outCtx) {
	handleStatus(hipMemcpy(nv12.data(), outCtx.cudaNv12ptr, nv12.size(), hipMemcpyDeviceToHost), "error getting nv12 data");
}


void cudaGetTransformedOutput(float* warpedData, const CudaData& core) {
	std::vector<float4> data(1ull * core.w * core.h);
	size_t wbytes = core.w * sizeof(float4);
	hipMemcpy2D(data.data(), wbytes, out.warped, core.strideFloat4, wbytes, core.h, hipMemcpyDefault);

	for (int i = 0; i < core.w * core.h; i++) {
		warpedData[i] = data[i].x;
		warpedData[i + core.w * core.h] = data[i].y;
		warpedData[i + core.w * core.h * 2] = data[i].z;
	}
}

void cudaGetPyramid(float* pyramid, size_t idx, const CudaData& core) {
	size_t pyrIdx = idx % core.pyramidCount;
	float* devptr = d_pyrData + pyrIdx * core.pyramidRowCount * 3 * core.strideFloatN;
	size_t wbytes = core.w * sizeof(float);
	hipMemcpy2D(pyramid, wbytes, devptr, core.strideFloat, wbytes, core.pyramidRowCount * 3ull, hipMemcpyDefault);
}

ImageYuv cudaGetInput(int64_t index, const CudaData& core) {
	ImageYuv out(core.h, core.w, core.w);
	int64_t fr = index % core.bufferCount;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * 3 * core.h * core.strideChar;
	//copy 2D data without stride
	hipMemcpy2D(out.data(), out.w, yuvSrc, core.strideChar, out.w, 3ll * out.h, hipMemcpyDefault);
	return out;
}

void cudaGetCurrentInputFrame(ImagePPM& image, const CudaData& core, int64_t idx) {
	int fridx = idx % core.bufferCount;
	unsigned char* yuvSrc = d_yuvData + fridx * 3ull * core.h * core.strideChar;
	cu::yuv_to_rgb(yuvSrc, core.strideChar, d_rgb, core.w, core.w, core.h);
	handleStatus(hipMemcpy(image.data(), d_rgb, 3ull * core.w * core.h, hipMemcpyDefault), "error @progress input");
}

void cudaGetCurrentOutputFrame(ImagePPM& image, const CudaData& core) {
	cu::yuv_to_rgb(out.warped, core.strideFloat4N, d_rgb, core.w, core.w, core.h);
	handleStatus(hipMemcpy(image.data(), d_rgb, 3ull * core.w * core.h, hipMemcpyDefault), "error @progress output");
}


//----------------------------------
//-------- SYNCHRONIZE
//----------------------------------

void cudaSynchronize() {
	handleStatus(hipDeviceSynchronize(), "error @synchronize #10");
	handleStatus(hipGetLastError(), "error @synchronize #20");
}


//----------------------------------
//-------- SHUTDOWN
//----------------------------------

DebugData cudaShutdown(const CudaData& core) {
	//get debug data
	std::vector<double> outDebug(debugData.maxSize / sizeof(double));
	handleStatus(hipMemcpy(outDebug.data(), debugData.d_data, debugData.maxSize, hipMemcpyDeviceToHost), "error @shutdown #5 copy debug data");

	//get image of kernel timing values
	int siz = core.computeBlocks.x * core.computeBlocks.y;
	std::vector<KernelTimer> kernelTimer(siz);
	int h = (int) kernelTimer.size();
	int w = 8'000;
	ImageBGR kernelTimerImage(h, w);
	
	handleStatus(hipMemcpy(kernelTimer.data(), d_kernelTimer, sizeof(KernelTimer) * kernelTimer.size(), hipMemcpyDefault), "error @shutdown #110");

	auto fcnMin = [] (KernelTimer& kt1, KernelTimer& kt2) { return kt1.timeStart < kt2.timeStart; };
	auto minTime = std::min_element(kernelTimer.begin(), kernelTimer.end(), fcnMin);
	auto fcnMax = [] (KernelTimer& kt1, KernelTimer& kt2) { return kt1.timeStop < kt2.timeStop; };
	auto maxTime = std::max_element(kernelTimer.begin(), kernelTimer.end(), fcnMax);
	int64_t delta = maxTime->timeStop - minTime->timeStart;
	if (delta > 0) {
		double f = delta / (w - 1.0);
		for (int i = 0; i < core.computeBlocks.x * core.computeBlocks.y; i++) {
			KernelTimer& kt = kernelTimer[i];
			int t1 = int((kt.timeStart - minTime->timeStart) / f);
			int t2 = int((kt.timeStop - minTime->timeStart) / f);
			for (int k = t1; k <= t2; k++) {
				kernelTimerImage.at(0, i, k) = 255;
			}
		}
	}
	
	//delete device memory
	void* d_arr[] = { d_results, d_yuvOut, d_rgb, d_yuvData, d_yuvRows, d_yuvPlanes, 
		out.data, d_bufferH, d_bufferV, d_pyrData, d_pyrRows, 
		debugData.d_data, d_kernelTimer, d_interrupt, d_computed 
	};

	for (void* ptr : d_arr) {
		handleStatus(hipFree(ptr), "error @shutdown #10 shutting down memory");
	}

	//delete streams
	for (int i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamDestroy(cs[i]), "error @shutdown #20 shutting down streams");
	}

	//unregister memory
	handleStatus(hipHostUnregister(registeredMemPtr), "error @shutdown #30 unregister");

	//do not reset device while nvenc is still active
	//handleStatus(hipDeviceReset(), "error @shutdown #90", errorList);
	handleStatus(hipGetLastError(), "error @shutdown #100");

	return { outDebug, kernelTimerImage };
}
