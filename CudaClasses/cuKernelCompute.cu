#include "hip/hip_runtime.h"
/*
 * This file is part of CUVISTA - Cuda Video Stabilizer
 * Copyright (c) 2023 Rainer Bitschi cuvista@a1.net
 *
 * This program is free software : you can redistribute it and /or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < http://www.gnu.org/licenses/>.
 */

#include "cuDeshaker.cuh"

using uint = unsigned int;

__constant__ double wp0[] = { 1, 0, 0, 0, 1, 0, 0, 0, 1 };
__constant__ double eta0[] = { 0, 0, 1, 0, 0, 1 };
__constant__ CoreData d_core;

//interpolate value for matrix given by pointer-pointer and w, h and return result in last parameter
template <class T> __device__ void interp2(T** arr, int w, int h, double x, double y, double& out) {
	if (x < 0.0 || x > w - 1.0 || y < 0.0 || y > h - 1.0) {
		out = d_core.dnan;

	} else {
		double flx = floor(x), fly = floor(y);
		double dx = x - flx, dy = y - fly;
		int ix = (int) flx, iy = (int) fly;

		double f00 = arr[iy][ix];
		double f01 = dx == 0 ? f00 : arr[iy][ix + 1];
		double f10 = dy == 0 ? f00 : arr[iy + 1][ix];
		double f11 = dx == 0 || dy == 0 ? f00 : arr[iy + 1][ix + 1];
		out = (1 - dx) * (1 - dy) * f00 + (1 - dx) * dy * f10 + dx * (1 - dy) * f01 + dx * dy * f11;
	}
}

//compute displacement
//one block works one point in the image
__global__ void kernelCompute(DevicePointers devptr, PointResult* results, int64_t frameIdx, cu::DebugData debugData) {
	//size_t i = (blockIdx.y * gridDim.x + blockIdx.x) * timerCount; KERNEL_TIME(timestamps, i++);
	int& ir = d_core.ir;
	int& iw = d_core.iw;

	//allocate individual variables in shared memory
	extern __shared__ double shd[];
	double* ptr = shd;
	double* sd = ptr;		ptr += 6 * iw * iw; // 6 x iw*iw
	double* s = ptr;		ptr += 36; // 6 x 6
	double* g = ptr;		ptr += 36; // 6 x 6
	double* im = ptr;		ptr += iw * iw; // iw x iw
	double* jm = ptr;		ptr += iw * iw; // iw x iw
	double* delta = ptr;	ptr += iw * iw; // iw x iw
	double* wp = ptr;		ptr += 9; // 3 x 3
	double* dwp = ptr;		ptr += 9; // 3 x 3
	double* b = ptr;        ptr += 6; //b 6 doubles
	double* eta = ptr;      ptr += 6; //eta 6 doubles
	double* temp = ptr;     ptr += 6; //temp 6 doubles
	//array of double pointers to get rows in LU decomposition
	double** Apiv = (double**) (ptr); ptr += 6;

	const uint ci = threadIdx.x;	//0..warpSize/iw    column into image
	const uint cols = blockDim.x;	//columns that can be addressed in one warp
	const uint r = threadIdx.y;		//0..iw   row into image
	const int rir = r - ir;

	//init wp and dwp to identitiy
	if (r < 3 && ci < 3) {
		dwp[r * 3 + ci] = wp[r * 3 + ci] = wp0[r * 3 + ci];
	}

	uint ix0 = blockIdx.x;
	uint iy0 = blockIdx.y;
	//center point of image patch in this block
	int ym = iy0 + ir;
	int xm = ix0 + ir;
	PointResultType result = PointResultType::RUNNING;

	int z = d_core.zMax;
	for (; z >= d_core.zMin && result >= PointResultType::RUNNING; z--) {
		//dimensions for current pyramid level
		int wz = d_core.w >> z;
		int hz = d_core.h >> z;

		if (r < iw) {
			for (int c = ci; c < iw; c += cols) {
				//copy area of interest from previous frame
				im[r * iw + c] = devptr.Yprev[ym + rir][xm - ir + c];

				//build sd matrix [6 x iw*iw]
				double x = devptr.DXprev[ym - ir + c][xm + rir];
				double y = devptr.DYprev[ym - ir + c][xm + rir];
				int idx = r * iw + c;
				sd[idx] = x;				
				idx += iw * iw;
				sd[idx] = y;				
				idx += iw * iw;
				sd[idx] = x * rir;			
				idx += iw * iw;
				sd[idx] = y * rir;			
				idx += iw * iw;
				sd[idx] = x * (c - ir);		
				idx += iw * iw;
				sd[idx] = y * (c - ir);
			}
		}
		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 49, sd);

		//S = sd * sd' [6 x 6]
		if (r < 6) {
			for (int c = r + ci; c < 6; c += cols) {
				//compute only upper triangle
				s[r * 6 + c] = 0.0;
				for (int i = 0; i < iw * iw; i++) {
					s[r * 6 + c] += sd[r * iw * iw + i] * sd[c * iw * iw + i];
				}
				//copy symmetric value
				s[c * 6 + r] = s[r * 6 + c];
			}
		}
		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 6, s);
		//KERNEL_TIME(timestamps, i++);

		//compute norm before starting inverse, s will be overwritten
		double ns = norm1(s, 6, 6, temp);
		//invert sd -> g [6 x 6]
		luinv(Apiv, s, temp, g, 6, r, ci, cols);
		//compute reciprocal condition, see if result is valid
		double ng = norm1(g, 6, 6, temp);
		double rcond = 1 / (ns * ng);
		result = (isnan(rcond) || rcond < d_core.deps) ? PointResultType::FAIL_SINGULAR : PointResultType::RUNNING;

		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 6, g); //----------------------------
		//KERNEL_TIME(timestamps, i++);

		//init loop limit counter
		int iter = 0;
		//init error measure to stop loop
		double bestErr = d_core.dmax;
		//main loop to find transformation of image patch
		while (result == PointResultType::RUNNING) {
			//interpolate image patch
			if (r < iw) {
				for (int c = ci; c < iw; c += cols) {
					int x = c - ir;
					double ix = xm + x * wp[0] + rir * wp[3] + wp[2];
					double iy = ym + x * wp[1] + rir * wp[4] + wp[5];
					interp2(devptr.Ycur, wz, hz, ix, iy, jm[r * iw + c]);
				}
			}

			//compute difference between image patches
			if (r < iw) {
				for (int c = ci; c < iw; c += cols) {
					delta[c * iw + r] = im[r * iw + c] - jm[r * iw + c]; //store delta in transposed order
				}
			}

			//eta = g.times(sd.times(delta.flatToCol())) [6 x 1]
			if (r < 6 && ci == 0) {
				//init eta to [0 0 1 0 0 1]
				eta[r] = eta0[r];
				//init b to [0 0 0 0 0 0]
				b[r] = 0.0;
				//sd * delta
				for (double* sdptr = sd + r * iw * iw, *deltaptr = delta; deltaptr != delta + iw * iw; sdptr++, deltaptr++) {
					b[r] += (*sdptr) * (*deltaptr);
				}
				//g * (sd * delta)
				for (double* gptr = g + r * 6, *bptr = b; bptr != b + 6; gptr++, bptr++) {
					eta[r] += (*gptr) * (*bptr);
				}
			}

			//update transform matrix
			if (r < 2 && ci == 0) {
				//update wp to dwp
				dwp[r * 3 + 0] = wp[r * 3] * eta[2] + wp[r * 3 + 1] * eta[4];
				dwp[r * 3 + 1] = wp[r * 3] * eta[3] + wp[r * 3 + 1] * eta[5];
				dwp[r * 3 + 2] = wp[r * 3] * eta[0] + wp[r * 3 + 1] * eta[1] + wp[r * 3 + 2];

				//update wp
				wp[r * 3 + 0] = dwp[r * 3];
				wp[r * 3 + 1] = dwp[r * 3 + 1];
				wp[r * 3 + 2] = dwp[r * 3 + 2];
			}
			//if (frameIdx == 1 && ix0 == 27 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 3, 3, wp); //--------------------------

			//analyse result, decide on continuing loop
			double err = eta[0] * eta[0] + eta[1] * eta[1];
			if (isnan(err)) result = PointResultType::FAIL_ETA_NAN; //leave loop with fail
			if (err < d_core.compMaxTol) result = PointResultType::SUCCESS_ABSOLUTE_ERR; //leave loop with success
			if (fabs(err - bestErr) / bestErr < d_core.compMaxTol * d_core.compMaxTol) result = PointResultType::SUCCESS_STABLE_ITER; //leave with success
			if (err < bestErr) bestErr = err;
			iter++;
			if (iter == d_core.compMaxIter && result == PointResultType::RUNNING) result = PointResultType::FAIL_ITERATIONS; //leave with fail
		}
		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 1, eta);

		//displacement * 2 for next level
		if (r == 0 && ci == 0) wp[2] *= 2.0;
		if (r == 1 && ci == 0) wp[5] *= 2.0;
		
		//center of integration window on next level
		xm *= 2;
		ym *= 2;

		//update pointers into pyramid for next higher level, move up the number of rows
		int rowsToMove = d_core.h >> (z - 1);
		devptr.movePosition(-rowsToMove);

		//KERNEL_TIME(timestamps, i++);
	}
	//for (int i = z; i >= d_core.zMin; i--) {
	//	KERNEL_TIME(timestamps, i++);
	//	KERNEL_TIME(timestamps, i++);
	//	KERNEL_TIME(timestamps, i++);
	//}

	if (cu::firstThread()) {
		//final displacement vector
		double u = wp[2];
		double v = wp[5];
		//bring values to level 0
		while (z < 0) { xm /= 2; ym /= 2; u /= 2; v /= 2; z++; }
		while (z > 0) { xm *= 2; ym *= 2; u *= 2; v *= 2; z--; }
		//index into results array
		size_t idx = iy0 * gridDim.x + ix0;
		//store results object
		results[idx] = { idx, ix0, iy0, xm, ym, xm - d_core.w / 2, ym - d_core.h / 2, u, v, result };
	}
	//KERNEL_TIME(timestamps, i++);

}

void kernelComputeCall(kernelParam param, DevicePointers pointers, PointResult* d_results, int64_t frameIdx, cu::DebugData debugData) {
	kernelCompute << <param.blk, param.thr, param.shdBytes, param.stream >> > (pointers, d_results, frameIdx, debugData);
}

void computeInit(const CoreData& core) {
	//copy core struct to device
	const void* ptr = &d_core;
	hipMemcpyToSymbol(HIP_SYMBOL(ptr), &core, sizeof(core));
}
