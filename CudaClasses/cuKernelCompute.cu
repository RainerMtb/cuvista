#include "hip/hip_runtime.h"
/*
 * This file is part of CUVISTA - Cuda Video Stabilizer
 * Copyright (c) 2023 Rainer Bitschi cuvista@a1.net
 *
 * This program is free software : you can redistribute it and /or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < http://www.gnu.org/licenses/>.
 */

#include "cuDeshaker.cuh"

using uint = unsigned int;

struct ArrayIndex {
	int r, c;
};

//map from thread index to S upper triangle
__constant__ ArrayIndex sidx[] = {
	{0,0}, {0,1}, {0,2}, {0,3}, {0,4}, {0,5},
	       {1,1}, {1,2}, {1,3}, {1,4}, {1,5},
	              {2,2}, {2,3}, {2,4}, {2,5},
	                     {3,3}, {3,4}, {3,5},
	                            {4,4}, {4,5},
	                                   {5,5},
};

//list of possible results of one compute iteration
__constant__ PointResultType resultTypes[] = {
	PointResultType::RUNNING, 
	PointResultType::FAIL_ETA_NAN, 
	PointResultType::SUCCESS_ABSOLUTE_ERR, 
	PointResultType::SUCCESS_STABLE_ITER,
	PointResultType::FAIL_ITERATIONS,
};

//initial values for wp
__constant__ double wp0[] = { 1, 0, 0, 0, 1, 0, 0, 0, 1 };
//initial values for eta
__constant__ double eta0[] = { 0, 0, 1, 0, 0, 1 };

//parameter structure
__constant__ CoreData d_core;

//forward declaration to prevent underline
template<class T> __device__ T tex2D(hipTextureObject_t tex, float x, float y);

//compute displacement
//one cuda block works one point in the image using one warp
__global__ void kernelCompute(ComputeTextures tex, PointResult* results, ComputeKernelParam param) {
	uint ix0 = blockIdx.x;
	uint iy0 = blockIdx.y;
	uint blockIndex = iy0 * gridDim.x + ix0;
	if (*param.d_interrupt || param.d_computed[blockIndex]) return;
	param.kernelTimestamps[blockIndex].start();

	int& ir = d_core.ir;
	int& iw = d_core.iw;

	//allocate individual variables in shared memory
	extern __shared__ double shd[];
	double* ptr = shd;
	double* sd = ptr;		ptr += 6 * iw * iw;  // 6 x iw*iw
	double* s = ptr;		ptr += 36;           // 6 x 6
	double* g = ptr;		ptr += 36;           // 6 x 6
	double* delta = ptr;	ptr += iw * iw;      // iw x iw
	double* wp = ptr;		ptr += 9;            // 3 x 3
	double* dwp = ptr;		ptr += 9;            // 3 x 3
	double* b = ptr;        ptr += 6;            // 6 doubles
	double* eta = ptr;      ptr += 6;            //eta 6 doubles
	double* temp = ptr;     ptr += 6;            //temp 6 doubles
	//array of double pointers to get rows in LU decomposition
	double** Apiv = (double**) (ptr);

	const int ci = threadIdx.x;	    //column into image
	const int cols = blockDim.x;	//columns that can be addressed in one warp
	const int r = threadIdx.y;		//row into image
	const int tidx = threadIdx.y * blockDim.x + threadIdx.x;

	//init wp and dwp to identitiy
	if (r < 3 && ci < 3) {
		dwp[r * 3 + ci] = wp[r * 3 + ci] = wp0[r * 3 + ci];
	}

	//center point of image patch in this block
	int ym = iy0 + ir;
	int xm = ix0 + ir;
	PointResultType result = PointResultType::RUNNING;

	//pyramid level to start at
	int z = d_core.zMax;
	//offset in rows to current pyramid level as texture spans one full pyramid
	int rowOffset = d_core.pyramidRowCount - (d_core.h >> z);

	for (; z >= d_core.zMin && result >= PointResultType::RUNNING; z--) {
		//dimensions for current pyramid level
		int wz = d_core.w >> z;
		int hz = d_core.h >> z;

		//build sd matrix [6 x iw*iw]
		if (r < iw) {
			for (int c = ci; c < iw; c += cols) {
				double x = tex2D<float>(tex.DXprev, xm - ir + r, rowOffset + ym - ir + c);
				double y = tex2D<float>(tex.DYprev, xm - ir + r, rowOffset + ym - ir + c);
				int idx = r * iw + c;
				sd[idx] = x;
				idx += iw * iw;
				sd[idx] = y;
				idx += iw * iw;
				sd[idx] = x * (r - ir);
				idx += iw * iw;
				sd[idx] = y * (r - ir);
				idx += iw * iw;
				sd[idx] = x * (c - ir);
				idx += iw * iw;
				sd[idx] = y * (c - ir);
			}
		}
		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 49, sd);

		//S = sd * sd' [6 x 6]
		//compute upper triangle and mirror value to write all values for S
		if (tidx < 21) {
			ArrayIndex ai = sidx[tidx]; //the value to compute in S
			double sval = 0.0;
			for (int i = 0; i < iw * iw; i++) {
				sval += sd[ai.r * iw * iw + i] * sd[ai.c * iw * iw + i];
			}
			//copy symmetric value
			s[ai.c * 6 + ai.r] = s[ai.r * 6 + ai.c] = sval;
		}
		//if (frameIdx == 1 && ix0 == 20 && iy0 == 20 && cu::firstThread()) cu::storeDebugData(debugData, 6, 6, s);

		//compute norm before starting inverse, s will be overwritten
		double ns = norm1(s, 6, 6, temp);
		//invert sd -> g [6 x 6]
		luinv(Apiv, s, temp, g, 6, r, ci, cols);
		//compute reciprocal condition, see if result is valid
		double ng = norm1(g, 6, 6, temp);
		double rcond = 1 / (ns * ng);
		result = (isnan(rcond) || rcond < d_core.deps) ? PointResultType::FAIL_SINGULAR : PointResultType::RUNNING;

		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 6, g);

		//init loop limit counter
		int iter = 0;
		//init error measure to stop loop
		double bestErr = d_core.dmax;
		//main loop to find transformation of image patch
		while (result == PointResultType::RUNNING) {
			//interpolate image patch
			if (r < iw) {
				for (int c = ci; c < iw; c += cols) {
					int x = c - ir;
					double ix = xm + x * wp[0] + (r - ir) * wp[3] + wp[2];
					double iy = ym + x * wp[1] + (r - ir) * wp[4] + wp[5];

					//compute difference between image patches
					//store delta in transposed order [c * iw + r]
					if (ix < 0.0 || ix > wz - 1.0 || iy < 0.0 || iy > hz - 1.0) {
						delta[c * iw + r] = d_core.dnan;

					} else {
						double im = tex2D<float>(tex.Yprev, xm - ir + c, rowOffset + ym + r - ir);

						double flx = floor(ix), fly = floor(iy);
						double dx = ix - flx, dy = iy - fly;
						int x0 = (int) flx, y0 = (int) fly;

						double f00 = tex2D<float>(tex.Ycur, x0, rowOffset + y0);
						double f01 = tex2D<float>(tex.Ycur, x0 + 1, rowOffset + y0);
						double f10 = tex2D<float>(tex.Ycur, x0, rowOffset + y0 + 1);
						double f11 = tex2D<float>(tex.Ycur, x0 + 1, rowOffset + y0 + 1);
						double jm = (1.0 - dx) * (1.0 - dy) * f00 + (1.0 - dx) * dy * f10 + dx * (1.0 - dy) * f01 + dx * dy * f11;

						delta[c * iw + r] = im - jm;
					}
				}
			}

			//eta = g.times(sd.times(delta.flatToCol())) [6 x 1]
			if (r < 6 && ci == 0) {
				//init eta to [0 0 1 0 0 1]
				eta[r] = eta0[r];
				//init b to [0 0 0 0 0 0]
				double bval = 0.0;
				//sd * delta_flat
				for (double* sdptr = sd + r * iw * iw, *deltaptr = delta; deltaptr != delta + iw * iw; sdptr++, deltaptr++) {
					bval += (*sdptr) * (*deltaptr);
				}
				b[r] = bval;
				//g * (sd * delta)
				for (double* gptr = g + r * 6, *bptr = b; bptr != b + 6; gptr++, bptr++) {
					eta[r] += (*gptr) * (*bptr);
				}
			}

			//update transform matrix
			if (r < 2 && ci == 0) {
				//update wp to dwp
				dwp[r * 3 + 0] = wp[r * 3] * eta[2] + wp[r * 3 + 1] * eta[4];
				dwp[r * 3 + 1] = wp[r * 3] * eta[3] + wp[r * 3 + 1] * eta[5];
				dwp[r * 3 + 2] = wp[r * 3] * eta[0] + wp[r * 3 + 1] * eta[1] + wp[r * 3 + 2];

				//update wp
				wp[r * 3 + 0] = dwp[r * 3];
				wp[r * 3 + 1] = dwp[r * 3 + 1];
				wp[r * 3 + 2] = dwp[r * 3 + 2];
			}
			//if (frameIdx == 1 && ix0 == 27 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 3, 3, wp); //--------------------------

			//analyse result, decide on continuing loop
			double err = eta[0] * eta[0] + eta[1] * eta[1];
			int typeIndex = 0;
			typeIndex += (int) isnan(err) * 1; //leave loop with fail message FAIL_ETA_NAN
			typeIndex += (int) (err < d_core.compMaxTol) * 2; //leave loop with success SUCCESS_ABSOLUTE_ERR
			typeIndex += (int) (fabs(err - bestErr) / bestErr < d_core.compMaxTol * d_core.compMaxTol) * 3; //SUCCESS_STABLE_ITER
			result = resultTypes[typeIndex];

			bestErr = min(err, bestErr);
			iter++;

			//typeIndex = (int) (iter == d_core.compMaxIter && result == PointResultType::RUNNING) * 4; //FAIL_ITERATIONS
			//result = resultTypes[typeIndex];
			if (iter == d_core.compMaxIter && result == PointResultType::RUNNING) {
				result = PointResultType::FAIL_ITERATIONS; //leave with fail
			}
		}
		//if (frameIdx == 1 && ix0 == 63 && iy0 == 1 && cu::firstThread()) cu::storeDebugData(debugData, 6, 1, eta);

		//displacement * 2 for next level
		if (r == 0 && ci == 0) wp[2] *= 2.0;
		if (r == 1 && ci == 0) wp[5] *= 2.0;

		//center of integration window on next level
		xm *= 2;
		ym *= 2;

		//new texture row offset
		int delta = d_core.h >> (z - 1);
		rowOffset -= delta;
	}

	if (cu::firstThread()) {
		//final displacement vector
		double u = wp[2];
		double v = wp[5];
		//bring values to level 0
		while (z < 0) { xm /= 2; ym /= 2; u /= 2; v /= 2; z++; }
		while (z > 0) { xm *= 2; ym *= 2; u *= 2; v *= 2; z--; }
		//index into results array
		size_t idx = iy0 * gridDim.x + ix0;
		//store results object
		results[idx] = { idx, ix0, iy0, xm, ym, xm - d_core.w / 2, ym - d_core.h / 2, u, v, result };
	}

	param.kernelTimestamps[blockIndex].stop();
	param.d_computed[blockIndex] = 1;
}

void kernelComputeCall(ComputeKernelParam param, ComputeTextures& tex, PointResult* d_results) {
	dim3 blk(param.blk.x, param.blk.y);
	dim3 thr(param.thr.x, param.thr.y);
	kernelCompute<<<blk, thr, param.shdBytes, param.stream>>> (tex, d_results, param);
}

void computeInit(const CoreData& core) {
	//copy core struct to device
	const void* ptr = &d_core;
	hipMemcpyToSymbol(HIP_SYMBOL(ptr), &core, sizeof(core));
}
