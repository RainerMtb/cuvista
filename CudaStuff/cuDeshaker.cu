#include "hip/hip_runtime.h"
/*
 * This file is part of CUVISTA - Cuda Video Stabilizer
 * Copyright (c) 2023 Rainer Bitschi cuvista@a1.net
 *
 * This program is free software : you can redistribute it and /or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < http://www.gnu.org/licenses/>.
 */

#include "cuDeshaker.cuh"
#include "cuNPP.cuh"
#include "Image.hpp"

#include <algorithm>
#include <fstream>

//parameter structure
//all values must be initialized to be used as __constant__ variable in device code, no constructor calls
__constant__ CoreData d_core;
CudaData cudaData;

//-------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------- HOST CODE ------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------

static void handleStatus(hipError_t status, std::string&& title) {
	if (status != hipSuccess) {
		errorLogger().logError(title + ": " + hipGetErrorString(status));
	}
}

static hipTextureObject_t prepareComputeTexture(float* src, int w, int h, int pitch) {
	hipResourceDesc resDesc {};
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = src;
	resDesc.res.pitch2D.width = w;
	resDesc.res.pitch2D.height = h;
	resDesc.res.pitch2D.pitchInBytes = pitch;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();

	// Specify texture object parameters
	hipTextureDesc texDesc {};
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;

	hipTextureObject_t texObj;
	handleStatus(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "error @compute 10");
	return texObj;
}

void ComputeTextures::create(int64_t idx, int64_t idxPrev, const CoreData& core, float* pyrBase) {
	size_t pyramidSize = 1ull * core.pyramidRowCount * cudaData.strideFloatN; //size of one full pyramid in elements
	float* ptrPrev = pyrBase + pyramidSize * idxPrev;
	Y[0] = prepareComputeTexture(ptrPrev, core.w, core.pyramidRowCount, cudaData.strideFloat);
	float* ptrCur = pyrBase + pyramidSize * idx;
	Y[1] = prepareComputeTexture(ptrCur, core.w, core.pyramidRowCount, cudaData.strideFloat);
}

void ComputeTextures::destroy() const {
	hipDestroyTextureObject(Y[0]);
	hipDestroyTextureObject(Y[1]);
}

//allocate cuda memory and store pointers
template <class T> void allocSafe(T* ptr, size_t size) {
	handleStatus(hipMalloc(ptr, size), "error @init allocating memory");
}

template <class T> void allocDeviceIndices(T*** indexArray, T* srcptr, size_t offset, size_t count) {
	std::vector<T*> idxarr(count);
	size_t siz = sizeof(T*) * count;
	for (size_t i = 0; i < count; i++) idxarr[i] = srcptr + i * offset;
	allocSafe(indexArray, siz);
	handleStatus(hipMemcpy(*indexArray, idxarr.data(), siz, hipMemcpyDefault), "error @init copy");
}

bool checkKernelParameters(int3 threads, int3 blocks, size_t shdsize, const hipDeviceProp_t& cudaProps) {
	bool out = true;
	out &= threads.x <= cudaProps.maxThreadsDim[0];
	out &= threads.y <= cudaProps.maxThreadsDim[1];
	out &= threads.z <= cudaProps.maxThreadsDim[2];
	out &= blocks.x <= cudaProps.maxGridSize[0];
	out &= blocks.y <= cudaProps.maxGridSize[1];
	out &= blocks.z <= cudaProps.maxGridSize[2];
	out &= shdsize <= cudaProps.sharedMemPerBlock;
	out &= threads.x * threads.y * threads.z <= cudaProps.maxThreadsPerBlock;
	return out;
}

bool checkKernelParameters(int3 threads, int3 blocks, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(threads, blocks, 0, cudaProps);
}

bool checkKernelParameters(const CudaData& core, const hipDeviceProp_t& cudaProps) {
	return checkKernelParameters(core.computeThreads, core.computeBlocks, core.computeSharedMem, cudaProps);
}

//write data from device pointer to file for debugging
template <class T> void writeDeviceDataToFile(const T* devData, size_t h, size_t wCount, size_t stride, const std::string& path) {
	hipDeviceSynchronize();
	std::vector<T> hostData(h * wCount);
	hipMemcpy2D(hostData.data(), sizeof(T) * wCount, devData, sizeof(T) * stride, sizeof(T) * wCount, h, hipMemcpyDeviceToHost);
	std::ofstream file(path, std::ios::binary);
	file.write(reinterpret_cast<char*>(&h), sizeof(size_t));
	file.write(reinterpret_cast<char*>(&wCount), sizeof(size_t));
	size_t sizT = sizeof(T);
	file.write(reinterpret_cast<char*>(&sizT), sizeof(size_t));
	file.write(reinterpret_cast<char*>(hostData.data()), hostData.size() * sizeof(T));
}

//write string into image given by device pointer
void writeText(const std::string& text, int x0, int y0, int scaleX, int scaleY, float* deviceData, const CoreData& core) {
	//create Image<float>
	int imh = 10 * scaleY;
	int siz = imh * cudaData.strideFloat;
	im::ImageBase<float> im(imh, core.w, cudaData.strideFloatN, 3);

	//copy three horizontal stripes into host memory
	for (size_t z = 0; z < 3; z++) {
		float* src = deviceData + (y0 + z * core.h) * cudaData.strideFloatN;
		float* dst = im.plane(z);
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}

	//write text
	im.writeText(text, x0, 0, scaleX, scaleY, im::TextAlign::TOP_LEFT); //write into host memory

	//copy YUV planes back into device memory
	for (size_t z = 0; z < 3; z++) {
		float* src = im.plane(z);
		float* dst = deviceData + (y0 + z * core.h) * cudaData.strideFloatN;
		hipMemcpy(dst, src, siz, hipMemcpyDefault);
	}
}

//----------------------------------
//-------- INIT --------------------
//----------------------------------

//check for cuda runtime installation, this only needs link to cudart_static.lib
CudaProbeResult cudaProbeRuntime() {
	CudaProbeResult out;
	//absence of cuda will report error "CUDA driver is insufficient for CUDA runtime version"
	hipRuntimeGetVersion(&out.runtimeVersion);
	hipDriverGetVersion(&out.driverVersion);

	//if we found a proper cuda installation, ask for list of devices
	int deviceCount = 0;
	if (out.driverVersion > 0) {
		handleStatus(hipGetDeviceCount(&deviceCount), "error probing cuda devices");
		for (int i = 0; i < deviceCount; i++) {
			hipDeviceProp_t devProp;
			handleStatus(hipGetDeviceProperties(&devProp, i), "error getting device properties");
			out.props.push_back(devProp);
		}

		//query npp version numbers, this loads nvcuda.dll
		//const NppLibraryVersion* libVer = nppGetLibVersion(); //nppc.lib
		//cudaInfo.nppMajor = libVer->major;
		//cudaInfo.nppMinor = libVer->minor;
		//cudaInfo.nppBuild = libVer->build;
	}
	return out;
}


//----------------------------------
//-------- CLASS DEFINITION --------
//----------------------------------


CudaExecutor::CudaExecutor(CoreData& data, DeviceInfoBase& deviceInfo, MovieFrame& frame, ThreadPoolBase& pool) :
	FrameExecutor(data, deviceInfo, frame, pool) {}


void CudaExecutor::cudaInit(CoreData& core, int devIdx, const hipDeviceProp_t& prop, ImageYuv& yuvFrame) {
	//copy device prop structure
	props = prop;

	//pin memory of transfer object
	registeredMemPtr = yuvFrame.data();
	handleStatus(hipHostRegister(registeredMemPtr, yuvFrame.sizeInBytes(), hipHostRegisterDefault), "error @init #2");

	const size_t h = core.h;
	const size_t w = core.w;

	handleStatus(hipSetDevice(devIdx), "cannot set device");
	//hipFree(0); //seems necessary in order to get a context later;

	//sum up required shared memory for compute kernel
	int doublesCount = 0
		+ core.iw * core.iw * 6   //sd
		+ core.iw * core.iw * 1	  //delta
		+ 6 * 6		//S
		+ 6 * 6     //g
		+ 3 * 3  	//wp
		+ 3 * 3     //dwp;
		+ 6 * 1		//b
		+ 6 * 1     //eta
		+ 6 * 1		//temp
		;
	cudaData.computeSharedMem = 0
		+ doublesCount * sizeof(double)   //number of double values in shared memory
		+ 6 * sizeof(double*)             //additional double pointers
		;

	//compute kernel configuration
	cudaData.computeBlocks = { core.ixCount, core.iyCount };
	int rows = std::max(core.iw, 6);
	int ws = prop.warpSize;
	cudaData.computeThreads = { ws / rows, rows };

	//determine memory requirements
	//size_t texAlign = prop.texturePitchAlignment;
	size_t pitch = 0;
	uchar* d_ptr8;
	hipMallocPitch(&d_ptr8, &pitch, w, 1);
	cudaData.strideChar = (int) pitch;
	hipFree(d_ptr8);

	float* d_ptr32;
	hipMallocPitch(&d_ptr32, &pitch, w * sizeof(float), 1);
	cudaData.strideFloat = (int) pitch;
	cudaData.strideFloatN = cudaData.strideFloat / sizeof(float);
	hipFree(d_ptr32);

	float4* d_ptr128;
	hipMallocPitch(&d_ptr128, &pitch, w * sizeof(float4), 1);
	cudaData.strideFloat4 = (int) pitch;
	cudaData.strideFloat4N = cudaData.strideFloat4 / sizeof(float4);
	hipFree(d_ptr128);

	//compute required heap size
	size_t frameSize8 = 3ull * cudaData.strideChar * h;  //bytes for yuv444 images
	size_t heapRequired = 0;
	heapRequired += frameSize8 * core.bufferCount;       //yuv input storage
	heapRequired += frameSize8;						     //yuv out
	heapRequired += frameSize8;						     //rgb out
	heapRequired += 2ull * cudaData.strideFloat * h;     //filter buffers
	heapRequired += cudaData.strideFloat * h * core.pyramidLevels * core.pyramidCount;     //pyramid of Y frames
	heapRequired += cudaData.strideFloat4 * h * cudaData.outBufferCount;                   //output buffer in floats
	heapRequired += sizeof(CudaPointResult) * core.resultCount;                            //array of results structure
	heapRequired += 10ull * 1024 * 1024;

	//set memory limit
	size_t heap = 0;
	handleStatus(hipDeviceGetLimit(&heap, hipLimitMallocHeapSize), "error @init #10");
	if (heapRequired < heap) {
		handleStatus(hipDeviceSetLimit(hipLimitMallocHeapSize, heapRequired), "error @init #20");
	}

	size_t yuvRowCount = 3ull * h * core.bufferCount;
	size_t memtotal, memfree1, memfree2;
	handleStatus(hipMemGetInfo(&memfree1, &memtotal), "error @init #30");

	//allocate debug storage
	allocSafe(&debugData.d_data, debugData.maxSize);
	handleStatus(hipMemset(debugData.d_data, 0, debugData.maxSize), "error @init #32");

	//allocate frameResult arrays
	allocSafe(&d_results, sizeof(CudaPointResult) * core.resultCount);
	h_results = new CudaPointResult[core.resultCount];

	//allocate output yuv array
	allocSafe(&d_yuvOut, frameSize8);
	allocSafe(&d_rgba, 4ull * w * h);

	//allocate memory for yuv input data in char format [0..255]
	allocSafe(&d_yuvData, frameSize8 * core.bufferCount);
	allocDeviceIndices(&d_yuvRows, d_yuvData, cudaData.strideChar, yuvRowCount);
	allocDeviceIndices(&d_yuvPlanes, d_yuvRows, h, core.bufferCount * 3ull);
	frameIndizes.assign(core.bufferCount, -1);

	//check pyramid indizes
	pyramidIndizes.assign(core.pyramidCount, -1);

	//allocate float buffers
	allocSafe(&out.data, cudaData.strideFloat4 * h * cudaData.outBufferCount);
	//name individual parts for convenience
	size_t outSize = h * cudaData.strideFloat4N;
	out.start = out.data;
	out.warped = out.start + outSize;
	out.filterH = out.warped + outSize;
	out.filterV = out.filterH + outSize;
	out.final = out.filterV + outSize;
	out.background = out.final + outSize;

	//float filter buffers
	allocSafe(&d_bufferH, cudaData.strideFloat * h);
	allocSafe(&d_bufferV, cudaData.strideFloat * h);

	//initialize background color in output buffer
	float4 bgval = { core.bgcolorYuv[0], core.bgcolorYuv[1], core.bgcolorYuv[2] };
	std::vector<float4> bg(w * h, bgval);
	//write to static background
	size_t siz = w * sizeof(float4);
	handleStatus(hipMemcpy2D(out.background, cudaData.strideFloat4, bg.data(), siz, siz, h, hipMemcpyDefault), "error @init 60");
	//write to first image
	handleStatus(hipMemcpy2D(out.warped, cudaData.strideFloat4, bg.data(), siz, siz, h, hipMemcpyDefault), "error @init 61");

	//allocate image pyramids, all the same strided width but increasingly shorter
	//number of rows through all three pyramids, Y, DX, DY
	size_t pyrTotalRows = 1ull * core.pyramidRowCount * core.pyramidCount;
	allocSafe(&d_pyrData, cudaData.strideFloat * pyrTotalRows);
	allocDeviceIndices(&d_pyrRows, d_pyrData, cudaData.strideFloatN, pyrTotalRows);

	//set up cuda streams
	cs.assign(2, 0);
	for (size_t i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamCreate(&cs[i]), "error @init #70");
	}

	//set up compute kernel
	allocSafe(&d_interrupt, 1);

	//memory statistics
	handleStatus(hipMemGetInfo(&memfree2, &memtotal), "error @init #80");
	cudaData.cudaMemTotal = memtotal;
	cudaData.cudaUsedMem = memfree1 - memfree2;

	//copy core struct to device
	const void* coreptr = &d_core;
	hipMemcpyToSymbol(HIP_SYMBOL(coreptr), &core, sizeof(core));

	//final error checks
	handleStatus(hipDeviceSynchronize(), "error @init #90");
	handleStatus(hipGetLastError(), "error @init #92");
}


//----------------------------------
//-------- READ
//----------------------------------

//copy yuv input to device
void CudaExecutor::inputData(int64_t frameIndex, const ImageYuv& inputFrame) {
	int64_t fr = frameIndex % mData.bufferCount;
	frameIndizes[fr] = frameIndex;
	size_t frameSizeBytes = 3ull * cudaData.strideChar * mData.h;
	unsigned char* d_frame = d_yuvData + fr * frameSizeBytes;
	handleStatus(hipMemcpy2D(d_frame, cudaData.strideChar, inputFrame.data(), inputFrame.stride, 
		mData.w, 3ull * mData.h, hipMemcpyDefault), "error @read #10");
	handleStatus(hipGetLastError(), "error @read #20");
}


//----------------------------------
//-------- PYRAMID
//----------------------------------

//create image pyramid
void CudaExecutor::createPyramid(int64_t frameIndex, AffineDataFloat trf, bool warp) {
	int w = mData.w;
	int h = mData.h;

	//get to the start of this yuv image
	int64_t frIdx = frameIndex % mData.bufferCount;
	unsigned char* yuvStart = d_yuvData + frIdx * cudaData.strideChar * h * 3;

	//get to the start of this pyramid
	int64_t pyrIdx = frameIndex % mData.pyramidCount;
	float* pyrStart = d_pyrData + pyrIdx * mData.pyramidRowCount * cudaData.strideFloatN;

	//to keep track of things
	pyramidIndizes[pyrIdx] = frameIndex;

	//first level of pyramid Y data
	if (warp) {
		cu::set_32f(pyrStart, cudaData.strideFloatN, w, h, 0);
		cu::scale_8u32f(yuvStart, cudaData.strideChar, d_bufferH, cudaData.strideFloatN, w, h);
		cu::warp_back_32f(d_bufferH, cudaData.strideFloatN, pyrStart, cudaData.strideFloatN, w, h, trf);

	} else {
		cu::scale_8u32f(yuvStart, cudaData.strideChar, pyrStart, cudaData.strideFloatN, w, h);
		cu::filter_32f_h(pyrStart, d_bufferH, cudaData.strideFloatN, w, h, 0);
		cu::filter_32f_v(d_bufferH, pyrStart, cudaData.strideFloatN, w, h, 0);
	}

	//lower levels
	float* src = pyrStart;
	float* dest = pyrStart + 1ull * cudaData.strideFloatN * h;
	for (int z = 1; z <= mData.zMax; z++) {
		cu::remap_downsize_32f(src, cudaData.strideFloatN, dest, cudaData.strideFloatN, w, h);
		w /= 2;
		h /= 2;
		src = dest;
		dest += 1ull * cudaData.strideFloatN * h;
	}

	handleStatus(hipGetLastError(), "error @pyramid");
}


//----------------------------------
//-------- COMPUTE
//----------------------------------

void CudaExecutor::computeStart(int64_t frameIndex, std::vector<PointResult>& results) {
	int64_t pyrIdx = frameIndex % mData.pyramidCount;
	int64_t pyrIdxPrev = (frameIndex - 1) % mData.pyramidCount;
	assert(frameIndex > 0 && pyramidIndizes[pyrIdx] == pyramidIndizes[pyrIdxPrev] + 1 && "wrong frames to compute"); 

	//prepare kernel
	assert(checkKernelParameters(cudaData, props) && "invalid kernel parameters");
	computeTexture.create(pyrIdx, pyrIdxPrev, mData, d_pyrData);

	//reset computed flags
	handleStatus(hipMemsetAsync(d_results, 0, sizeof(CudaPointResult) * mData.resultCount, cs[0]), "error @compute #20");

	//issue the call
	ComputeKernelParam param = { 
		debugData.d_data,
		debugData.maxSize,
		cudaData.computeBlocks,
		cudaData.computeThreads,
		cudaData.computeSharedMem,
		cs[0], 
		frameIndex, 
		d_interrupt
	};
	kernelComputeCall(param, computeTexture, d_results);

	//hipStreamQuery(cs[0]);
	handleStatus(hipGetLastError(), "error @compute #20");
}

void CudaExecutor::computeTerminate(int64_t frameIndex, std::vector<PointResult>& results) {
	//reset interrupt signal
	handleStatus(hipMemsetAsync(d_interrupt, 0, sizeof(char), cs[1]), "error @compute #50");

	//restart kernel
	ComputeKernelParam param = {
		debugData.d_data,
		debugData.maxSize,
		cudaData.computeBlocks,
		cudaData.computeThreads,
		cudaData.computeSharedMem,
		cs[0],
		frameIndex,
		d_interrupt
	};
	kernelComputeCall(param, computeTexture, d_results);

	//get results from device
	handleStatus(hipMemcpy(h_results, d_results, sizeof(CudaPointResult) * mData.resultCount, hipMemcpyDefault), "error @compute #100");

	//translate to host structure
	for (int i = 0; i < mData.resultCount; i++) {
		const CudaPointResult& hr = h_results[i];
		double x0 = hr.xm - mData.w / 2.0 + hr.u * hr.direction;
		double y0 = hr.ym - mData.h / 2.0 + hr.v * hr.direction;
		double fdir = 1.0 - 2.0 * hr.direction;
		results[i] = { hr.idx, hr.ix0, hr.iy0, x0, y0, hr.u * fdir, hr.v * fdir, hr.result, hr.z, hr.direction };
	}

	//shutdown
	computeTexture.destroy();
	handleStatus(hipGetLastError(), "error @compute #100");
}


//----------------------------------
//-------- OUTPUT
//----------------------------------

void CudaExecutor::outputData(int64_t frameIndex, AffineDataFloat trf) {
	//ConsoleTimer timer;
	//interrupt compute kernel
	handleStatus(hipMemsetAsync(d_interrupt, 1, sizeof(char), cs[1]), "error @output #10");

	int h = mData.h;
	int w = mData.w;
	int64_t fr = frameIndex % mData.bufferCount;
	assert(frameIndizes[fr] == frameIndex && "invalid frame in buffer");

	//size of all pixel data in bytes in yuv including padding
	size_t frameSize8 = 3ull * cudaData.strideChar * h;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * frameSize8;

	cu::scale_8u32f_3(yuvSrc, cudaData.strideChar, out.start, cudaData.strideFloat4N, w, h, cs[1]);
	//fill static background when requested
	if (mData.bgmode == BackgroundMode::COLOR) {
		cu::copy_32f_3(out.background, cudaData.strideFloat4N, out.warped, cudaData.strideFloat4N, w, h, cs[1]);
	}
	//warp input
	cu::warp_back_32f_3(out.start, cudaData.strideFloat4N, out.warped, cudaData.strideFloat4N, w, h, trf, cs[1]);
	//writeDeviceDataToFile(out.start, h, w, cudaData.strideFloat4N, "f:/cuda.dat");
	//first filter pass
	cu::filter_32f_h_3(out.warped, out.filterH, cudaData.strideFloat4N, w, h, cs[1]);
	//second filter pass
	cu::filter_32f_v_3(out.filterH, out.filterV, cudaData.strideFloat4N, w, h, cs[1]);
	//combine unsharp mask
	cu::unsharp_32f_3(out.warped, out.filterV, out.final, cudaData.strideFloat4N, w, h, cs[1]);
}

void CudaExecutor::getOutputYuv(int64_t frameIndex, ImageYuv& image) const {
	cu::outputHost(out.final, cudaData.strideFloat4N, d_yuvOut, cudaData.strideChar, mData.w, mData.h, cs[1]);
	for (int planeSize = cudaData.strideChar * mData.h, i = 0; i < 3; i++) {
		cu::copy_32f_3(d_yuvOut + i * planeSize, cudaData.strideChar, image.addr(i, 0, 0), image.strideInBytes(), mData.w, mData.h, cs[1]);
	}
	image.setIndex(frameIndex);
	handleStatus(hipStreamSynchronize(cs[1]), "error @output #90");
	handleStatus(hipGetLastError(), "error @output #91");
}

void CudaExecutor::getOutputImage(int64_t frameIndex, ImageBaseRgb& image) const {
	const std::vector<int>& idx = image.indexRgba();
	cu::yuv_to_rgba(out.final, cudaData.strideFloat4N, d_rgba, -1, mData.w, mData.h, { idx[0], idx[1], idx[2], idx[3] }, cs[1]);
	handleStatus(hipMemcpyAsync(image.plane(0), d_rgba, 4ull * mData.w * mData.h, hipMemcpyDefault, cs[1]), "error @output #94");
	image.setIndex(frameIndex);
	handleStatus(hipStreamSynchronize(cs[1]), "error @output #92");
	handleStatus(hipGetLastError(), "error @output #93");
}

void CudaExecutor::getOutputNvenc(int64_t frameIndex, ImageNV12& image, unsigned char* cudaNv12ptr) const {
	cu::outputNvenc(out.final, cudaData.strideFloat4N, cudaNv12ptr, image.stride, mData.w, mData.h, cs[1]);
	handleStatus(hipStreamSynchronize(cs[1]), "error @output #95");
	handleStatus(hipGetLastError(), "error @output #96");
}

void CudaExecutor::cudaGetTransformedOutput(float* warped) const {
	std::vector<float4> data(1ull * mData.w * mData.h);
	size_t wbytes = mData.w * sizeof(float4);
	handleStatus(hipMemcpy2D(data.data(), wbytes, out.warped, cudaData.strideFloat4, wbytes, mData.h, hipMemcpyDefault), "error @transformedOutput");

	for (int i = 0; i < mData.w * mData.h; i++) {
		warped[i] = data[i].x;
		warped[i + mData.w * mData.h] = data[i].y;
		warped[i + mData.w * mData.h * 2] = data[i].z;
	}
}

void CudaExecutor::cudaGetPyramid(int64_t frameIndex, float* data) const {
	int pyrIdx = frameIndex % mData.pyramidCount;
	float* devptr = d_pyrData + pyrIdx * mData.pyramidRowCount * cudaData.strideFloatN;
	size_t wbytes = mData.w * sizeof(float);

	handleStatus(hipMemcpy2D(data, wbytes, devptr, cudaData.strideFloat, wbytes, mData.pyramidRowCount, hipMemcpyDefault), "error @getPyramid");
}

void CudaExecutor::getInput(int64_t frameIndex, ImageYuv& image) const {
	int fr = frameIndex % mData.bufferCount;
	//start of input yuv data
	unsigned char* yuvSrc = d_yuvData + fr * 3 * mData.h * cudaData.strideChar;
	//copy 2D data without stride
	handleStatus(hipMemcpy2D(image.data(), image.stride, yuvSrc, cudaData.strideChar, image.w, 3ll * image.h, hipMemcpyDefault), "error @getInput");
}

void CudaExecutor::getInput(int64_t frameIndex, ImageRGBA& image) const {
	int fridx = frameIndex % mData.bufferCount;
	assert(frameIndizes[fridx] == frameIndex && "invalid frame in buffer");
	unsigned char* yuvSrc = d_yuvData + fridx * 3ull * mData.h * cudaData.strideChar;
	cu::yuv_to_rgba(yuvSrc, cudaData.strideChar, d_rgba, -1, mData.w, mData.h);
	handleStatus(hipMemcpy(image.plane(0), d_rgba, 4ull * mData.w * mData.h, hipMemcpyDefault), "error @progress input");
}

void CudaExecutor::getWarped(int64_t frameIndex, ImageRGBA& image) {
	int4 index = { 0, 1, 2, 3 };
	cu::yuv_to_rgba(out.warped, cudaData.strideFloat4N, d_rgba, -1, mData.w, mData.h, index);
	handleStatus(hipMemcpy(image.plane(0), d_rgba, 4ull * mData.w * mData.h, hipMemcpyDefault), "error @progress output");
}


void encodeNvData(const ImageNV12& image, unsigned char* nvencPtr) {
	handleStatus(hipMemcpy(nvencPtr, image.addr(0, 0, 0), image.sizeInBytes(), hipMemcpyHostToDevice), "error @simple encode #1 cannot copy to device");
}

void getNvData(std::vector<unsigned char>& nv12, unsigned char* cudaNv12ptr) {
	handleStatus(hipMemcpy(nv12.data(), cudaNv12ptr, nv12.size(), hipMemcpyDeviceToHost), "error getting nv12 data");
}

/*
void cudaSynchronize() {
	handleStatus(hipDeviceSynchronize(), "error @synchronize #10");
	handleStatus(hipGetLastError(), "error @synchronize #20");
}
*/


//----------------------------------
//-------- SHUTDOWN
//----------------------------------

void CudaExecutor::getDebugData(const CoreData& core, const std::string& imageFile, std::function<void(size_t, size_t, double*)> fcn) {
	std::vector<double> data(debugData.maxSize / sizeof(double));
	handleStatus(hipMemcpy(data.data(), debugData.d_data, debugData.maxSize, hipMemcpyDefault), "error @shutdown #5 copy debug data");

	double* ptr = data.data() + 1;
	double* ptrEnd = data.data() + size_t(data[0]) + 1;
	while (ptr != ptrEnd) {
		size_t h = (size_t) *ptr++;
		size_t w = (size_t) *ptr++;
		fcn(h, w, ptr);
		ptr += h * w;
	}

	//get image of kernel timing values
	int h = core.resultCount;
	int w = 6'000;
	ImageBGR kernelTimerImage = ImageBGR(h, w);
	auto fcnMin = [] (CudaPointResult& r1, CudaPointResult& r2) { return r1.timeStart < r2.timeStart; };
	auto minTime = std::min_element(h_results, h_results + core.resultCount, fcnMin);
	auto fcnMax = [] (CudaPointResult& r1, CudaPointResult& r2) { return r1.timeStop < r2.timeStop; };
	auto maxTime = std::max_element(h_results, h_results + core.resultCount, fcnMax);
	int64_t delta = maxTime->timeStop - minTime->timeStart;
	if (delta > 0) {
		double f = delta / (w - 1.0);
		for (int i = 0; i < h; i++) {
			CudaPointResult& r = h_results[i];
			int t1 = int((r.timeStart - minTime->timeStart) / f);
			int t2 = int((r.timeStop - minTime->timeStart) / f);
			for (int k = t1; k <= t2; k++) {
				kernelTimerImage.at(0, i, k) = 255;
			}
		}
	}
	kernelTimerImage.saveAsColorBMP(imageFile);
}

CudaExecutor::~CudaExecutor() {
	
	//delete device memory
	void* d_arr[] = { d_results, d_yuvOut, d_rgba, d_yuvData, d_yuvRows, d_yuvPlanes, 
		out.data, d_bufferH, d_bufferV, d_pyrData, d_pyrRows, 
		debugData.d_data, d_interrupt
	};

	for (void* ptr : d_arr) {
		handleStatus(hipFree(ptr), "error @shutdown #10 shutting down memory");
	}

	//delete streams
	for (int i = 0; i < cs.size(); i++) {
		handleStatus(hipStreamDestroy(cs[i]), "error @shutdown #20 shutting down streams");
	}

	//delete host memory
	delete[] h_results;

	//unregister memory
	handleStatus(hipHostUnregister(registeredMemPtr), "error @shutdown #30 unregister");

	//do not reset device while nvenc is still active
	//handleStatus(hipDeviceReset(), "error @shutdown #90", errorList);
	handleStatus(hipGetLastError(), "error @shutdown #100");
}
